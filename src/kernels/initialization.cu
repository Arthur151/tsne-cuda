#include "../include/kernels/initialization.h"

/******************************************************************************/
/*** initialize memory ********************************************************/
/******************************************************************************/


void tsnecuda::Initialize(tsnecuda::GpuOptions &gpu_opt, thrust::device_vector<int> &errd)
{
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(tsnecuda::bh::IntegrationKernel), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(tsnecuda::bh::ComputePijxQijKernel), hipFuncCachePreferShared);
    GpuErrorCheck(hipDeviceSynchronize());
}
