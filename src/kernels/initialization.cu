#include "hip/hip_runtime.h"
/*
Kernel to initialize the global variables
*/

#include "include/kernels/initialization.h"

__device__ volatile int stepd, bottomd, maxdepthd;
__device__ unsigned int blkcntd;
__device__ volatile float radiusd;

/******************************************************************************/
/*** initialize memory ********************************************************/
/******************************************************************************/

__global__ void tsnecuda::bh::InitializationKernel(int * __restrict errd)
{
    *errd = 0;
    stepd = -1;
    maxdepthd = 1;
    blkcntd = 0;
}

void tsnecuda::bh::Initialize(tsnecuda::GpuOptions &gpu_opt, thrust::device_vector<int> &errd) 
{
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(tsnecuda::bh::BoundingBoxKernel), hipFuncCachePreferShared);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(tsnecuda::bh::TreeBuildingKernel), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(tsnecuda::bh::ClearKernel1), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(tsnecuda::bh::ClearKernel2), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(tsnecuda::bh::SummarizationKernel), hipFuncCachePreferShared);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(tsnecuda::bh::SortKernel), hipFuncCachePreferL1);
    #ifdef __KEPLER__
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(tsnecuda::bh::ForceCalculationKernel), hipFuncCachePreferEqual);
    #else
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(tsnecuda::bh::ForceCalculationKernel), hipFuncCachePreferL1);
    #endif
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(tsnecuda::bh::IntegrationKernel), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(tsnecuda::bh::ComputePijxQijKernel), hipFuncCachePreferShared);
    
    tsnecuda::bh::InitializationKernel<<<1, 1>>>(thrust::raw_pointer_cast(errd.data()));
    GpuErrorCheck(hipDeviceSynchronize());
}

void tsnecuda::naive::Initialize() 
{
    // TODO: Add cache config sets for naive
}
