#include "include/kernels/initialization.h"

/******************************************************************************/
/*** initialize memory ********************************************************/
/******************************************************************************/


void tsnecuda::bh::Initialize(tsnecuda::GpuOptions &gpu_opt, thrust::device_vector<int> &errd)
{
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(tsnecuda::bh::IntegrationKernel), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(tsnecuda::bh::ComputePijxQijKernel), hipFuncCachePreferShared);
    GpuErrorCheck(hipDeviceSynchronize());
}

void tsnecuda::naive::Initialize()
{
    // TODO: Add cache config sets for naive
}
