#include "hip/hip_runtime.h"
/*
Kernel to initialize the global variables
*/

#include "kernels/include/initialization.h"


/******************************************************************************/
/*** initialize memory ********************************************************/
/******************************************************************************/

__global__ void tsnecuda::bh::InitializationKernel(int * __restrict errd)
{
    *errd = 0;
    stepd = -1;
    maxdepthd = 1;
    blkcntd = 0;
}

void tsnecuda::bh::Initialize(int * __restrict__ errd) 
{
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(tsnecuda::bh::BoundingBoxKernel), hipFuncCachePreferShared);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(tsnecuda::bh::TreeBuildingKernel), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(tsnecuda::bh::ClearKernel1), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(tsnecuda::bh::ClearKernel2), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(tsnecuda::bh::SummarizationKernel), hipFuncCachePreferShared);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(tsnecuda::bh::SortKernel), hipFuncCachePreferL1);
    #ifdef __KEPLER__
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(tsnecuda::bh::ForceCalculationKernel), hipFuncCachePreferEqual);
    #else
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(tsnecuda::bh::ForceCalculationKernel), hipFuncCachePreferL1);
    #endif
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(tsnecuda::bh::IntegrationKernel), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(tsnecuda::bh::ComputePijxQijKernel), hipFuncCachePreferShared);
    
    tsnecuda::bh::InitializationKernel<<<1, 1>>>(thrust::raw_pointer_cast(errl.data()));
    GpuErrorCheck(hipDeviceSynchronize());
}

void tsnecuda::naive::Initialize() 
{
    // TODO: Add cache config sets for naive
}
