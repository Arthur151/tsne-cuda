#include "hip/hip_runtime.h"
/*
        Summarize position and mass of cells in quad-tree.
*/

#include "tree_summary.h"

/******************************************************************************/
/*** compute center of mass ***************************************************/
/******************************************************************************/

__global__
__launch_bounds__(SUMMARY_THREADS, SUMMARY_BLOCKS)
void tsnecuda::bh::SummarizationKernel(
                               volatile int * __restrict cell_counts, 
                               volatile float * __restrict cell_mass, 
                               volatile float * __restrict x_pos_device, 
                               volatile float * __restrict y_pos_device,
                               const int * __restrict children,
                               const uint32_t num_nodes,
                               const uint32_t num_points) 
{
    register int i, j, k, ch, inc, cnt, bottom, flag;
    register float m, cm, px, py;
    __shared__ int child[SUMMARY_THREADS * 4];
    __shared__ float mass[SUMMARY_THREADS * 4];

    bottom = bottomd;
    inc = blockDim.x * gridDim.x;
    k = (bottom & (-WARPSIZE)) + threadIdx.x + blockIdx.x * blockDim.x;    // align to warp size
    if (k < bottom) k += inc;

    register int restart = k;
    for (j = 0; j < 5; j++) {    // wait-free pre-passes
        // iterate over all cells assigned to thread
        while (k <= num_nodes) {
            if (cell_mass[k] < 0.0f) {
                for (i = 0; i < 4; i++) {
                    ch = children[k*4+i];
                    child[i*SUMMARY_THREADS+threadIdx.x] = ch;    // cache children
                    if ((ch >= num_points) && ((mass[i*SUMMARY_THREADS+threadIdx.x] = cell_mass[ch]) < 0.0f)) {
                        break;
                    }
                }
                if (i == 4) {
                    // all children are ready
                    cm = 0.0f;
                    px = 0.0f;
                    py = 0.0f;
                    cnt = 0;
                    for (i = 0; i < 4; i++) {
                        ch = child[i*SUMMARY_THREADS+threadIdx.x];
                        if (ch >= 0) {
                            if (ch >= num_points) {    // count bodies (needed later)
                                m = mass[i*SUMMARY_THREADS+threadIdx.x];
                                cnt += cell_counts[ch];
                            } else {
                                m = cell_mass[ch];
                                cnt++;
                            }
                            // add child's contribution
                            cm += m;
                            px += x_pos_device[ch] * m;
                            py += y_pos_device[ch] * m;
                        }
                    }
                    cell_counts[k] = cnt;
                    m = 1.0f / cm;
                    x_pos_device[k] = px * m;
                    y_pos_device[k] = py * m;
                    __threadfence();    // make sure data are visible before setting mass
                    cell_mass[k] = cm;
                }
            }
            k += inc;    // move on to next cell
        }
        k = restart;
    }

    flag = 0;
    j = 0;
    // iterate over all cells assigned to thread
    while (k <= num_nodes) {
        if (cell_mass[k] >= 0.0f) {
            k += inc;
        } else {
            if (j == 0) {
                j = 4;
                for (i = 0; i < 4; i++) {
                    ch = children[k*4+i];
                    child[i*SUMMARY_THREADS+threadIdx.x] = ch;    // cache children
                    if ((ch < num_points) || ((mass[i*SUMMARY_THREADS+threadIdx.x] = cell_mass[ch]) >= 0.0f)) {
                        j--;
                    }
                }
            } else {
                j = 4;
                for (i = 0; i < 4; i++) {
                    ch = child[i*SUMMARY_THREADS+threadIdx.x];
                    if ((ch < num_points) || (mass[i*SUMMARY_THREADS+threadIdx.x] >= 0.0f) || ((mass[i*SUMMARY_THREADS+threadIdx.x] = cell_mass[ch]) >= 0.0f)) {
                        j--;
                    }
                }
            }

            if (j == 0) {
                // all children are ready
                cm = 0.0f;
                px = 0.0f;
                py = 0.0f;
                cnt = 0;
                for (i = 0; i < 4; i++) {
                    ch = child[i*SUMMARY_THREADS+threadIdx.x];
                    if (ch >= 0) {
                        if (ch >= num_points) {    // count bodies (needed later)
                            m = mass[i*SUMMARY_THREADS+threadIdx.x];
                            cnt += cell_counts[ch];
                        } else {
                            m = cell_mass[ch];
                            cnt++;
                        }
                        // add child's contribution
                        cm += m;
                        px += x_pos_device[ch] * m;
                        py += y_pos_device[ch] * m;
                    }
                }
                cell_counts[k] = cnt;
                m = 1.0f / cm;
                x_pos_device[k] = px * m;
                y_pos_device[k] = py * m;
                flag = 1;
            }
        }
        __syncthreads();    
        __threadfence();
        if (flag != 0) {
            cell_mass[k] = cm;
            k += inc;
            flag = 0;
        }
    }
}

void tsnecuda::bh::SummarizeTree(thrust::device_vector<int> &cell_counts,
                                 thrust::device_vector<int> &children,
                                 thrust::device_vector<float> &cell_mass,
                                 thrust::device_vector<float> &pts_device,
                                 const uint32_t num_nodes,
                                 const uint32_t num_points,
                                 const uint32_t num_blocks)
{
    tsnecuda::bh::SummarizationKernel<<<num_blocks * SUMMARY_BLOCKS, SUMMARY_THREADS>>>(
                                                    thrust::raw_pointer_cast(cell_counts.data()),
                                                    thrust::raw_pointer_cast(cell_mass.data()),
                                                    thrust::raw_pointer_cast(pts_device.data()),
                                                    thrust::raw_pointer_cast(pts_device.data() + num_nodes + 1),
                                                    thrust::raw_pointer_cast(children.data()),
                                                    num_nodes, num_points);
    GpuErrorCheck(hipDeviceSynchronize());
}
