#include "hip/hip_runtime.h"
// TODO: add copyright

/*
        Traverses the tree and calculates approximate repulsive forces via Barnes-Hut.

        t-SNE repulsive forces are given by qij*qijN(y_i - y_j). This also simultaneously
        calculates tne normalization constant N.

*/

#include "include/kernels/bh_rep_forces.h"

/******************************************************************************/
/*** compute force ************************************************************/
/******************************************************************************/

__global__
__launch_bounds__(REPULSIVE_FORCES_THREADS, REPULSIVE_FORCES_BLOCKS)
void tsnecuda::bh::ForceCalculationKernel(volatile int * __restrict__ errd,
                                          volatile float * __restrict__ x_vel_device,
                                          volatile float * __restrict__ y_vel_device,
                                          volatile float * __restrict__ normalization_vec_device,
                                          const int * __restrict__ cell_sorted,
                                          const int * __restrict__ children,
                                          const float * __restrict__ cell_mass,
                                          volatile float * __restrict__ x_pos_device,
                                          volatile float * __restrict__ y_pos_device,
                                          const float theta,
                                          const float epsilon,
                                          const uint32_t num_nodes,
                                          const uint32_t num_points)
{
    register int i, j, k, n, depth, base, sbase, diff, pd, nd;
    register float px, py, vx, vy, dx, dy, normsum, tmp, mult;
    __shared__ volatile int pos[MAXDEPTH * REPULSIVE_FORCES_THREADS/WARPSIZE], node[MAXDEPTH * REPULSIVE_FORCES_THREADS/WARPSIZE];
    __shared__ float dq[MAXDEPTH * REPULSIVE_FORCES_THREADS/WARPSIZE];

    if (0 == threadIdx.x) {
        dq[0] = (radiusd * radiusd) / (theta * theta); 
        for (i = 1; i < maxdepthd; i++) {
                dq[i] = dq[i - 1] * 0.25f; // radius is halved every level of tree so squared radius is quartered
                dq[i - 1] += epsilon;
        }
        dq[i - 1] += epsilon;

        if (maxdepthd > MAXDEPTH) {
            *errd = maxdepthd;
        }
    }
    __syncthreads();

    if (maxdepthd <= MAXDEPTH) {
        // figure out first thread in each warp (lane 0)
        base = threadIdx.x / WARPSIZE;
        sbase = base * WARPSIZE;
        j = base * MAXDEPTH;

        diff = threadIdx.x - sbase;
        // make multiple copies to avoid index calculations later
        if (diff < MAXDEPTH) {
            dq[diff+j] = dq[diff];
        }
        __syncthreads();
        __threadfence_block();

        // iterate over all bodies assigned to thread
        for (k = threadIdx.x + blockIdx.x * blockDim.x; k < nbodiesd; k += blockDim.x * gridDim.x) {
            i = cell_sorted[k];    // get permuted/sorted index
            // cache position info
            px = x_pos_device[i];
            py = y_pos_device[i];

            vx = 0.0f;
            vy = 0.0f;
            normsum = 0.0f;

            // initialize iteration stack, i.e., push root node onto stack
            depth = j;
            if (sbase == threadIdx.x) {
                pos[j] = 0;
                node[j] = nnodesd * 4;
            }

            do {
                // stack is not empty
                pd = pos[depth];
                nd = node[depth];
                while (pd < 4) {
                    // node on top of stack has more children to process
                    n = children[nd + pd];    // load child pointer
                    pd++;

                    if (n >= 0) {
                        dx = px - x_pos_device[n];
                        dy = py - y_pos_device[n];
                        tmp = dx*dx + dy*dy + epsilon; // distance squared plus small constant to prevent zeros
                        if ((n < nbodiesd) || __all_sync(__activemask(), tmp >= dq[depth])) {    // check if all threads agree that cell is far enough away (or is a body)
                            // from bhtsne - sptree.cpp
                            tmp = 1 / (1 + tmp);
                            mult = cell_mass[n] * tmp;
                            normsum += mult;
                            mult *= tmp;
                            vx += dx * mult;
                            vy += dy * mult;
                        } else {
                            // push cell onto stack
                            if (sbase == threadIdx.x) {    // maybe don't push and inc if last child
                                pos[depth] = pd;
                                node[depth] = nd;
                            }
                            depth++;
                            pd = 0;
                            nd = n * 4;
                        }
                    } else {
                        pd = 4;    // early out because all remaining children are also zero
                    }
                }
                depth--;    // done with this level
            } while (depth >= j);

            if (stepd >= 0) {
                // update velocity
                x_vel_device[i] += vx;
                y_vel_device[i] += vy;
                normalization_vec_device[i] = normsum - 1.0f; // subtract one for self computation (qii)
            }
        }
    }
}

void tsnecuda::bh::ComputeRepulsiveForces(thrust::device_vector<int> &errd,
                                          thrust::device_vector<float> &repulsive_forces,
                                          thrust::device_vector<float> &normalization_vec,
                                          thrust::device_vector<int> &cell_sorted,
                                          thrust::device_vector<int> &children,
                                          thrust::device_vector<float> &cell_mass,
                                          thrust::device_vector<float> &points,
                                          const float theta,
                                          const float epsilon,
                                          const uint32_t num_nodes,
                                          const uint32_t num_points,
                                          const uint32_t num_blocks)
{
    tsnecuda::bh::ForceCalculationKernel<<<num_blocks * REPULSIVE_FORCES_BLOCKS, REPULSIVE_FORCES_THREADS>>>(
                        thrust::raw_pointer_cast(errd.data()),
                        thrust::raw_pointer_cast(repulsive_forces.data()),
                        thrust::raw_pointer_cast(repulsive_forces.data() + num_nodes + 1),
                        thrust::raw_pointer_cast(normalization_vec.data()),
                        thrust::raw_pointer_cast(cell_sorted.data()),
                        thrust::raw_pointer_cast(children.data()),
                        thrust::raw_pointer_cast(cell_mass.data()),
                        thrust::raw_pointer_cast(points.data()),
                        thrust::raw_pointer_cast(points.data() + num_nodes + 1),
                        theta, epsilon, num_nodes, num_points);
    gpuErrchk(hipDeviceSynchronize());
}
