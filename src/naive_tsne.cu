#include "hip/hip_runtime.h"
/**
 * @brief Implementation of naive T-SNE
 * 
 * @file naive_tsne.cu
 * @author David Chan
 * @date 2018-04-04
 */

#include "naive_tsne.h"

struct func_inc_inv {
    __host__ __device__ float operator()(const float &x) const { return 1 / (x + 1); }
};

struct func_kl {
    __host__ __device__ float operator()(const float &x, const float &y) const { 
        return x == 0.0f ? 0.0f : x * (log(x) - log(y));
    }
};

struct func_exp {
    __host__ __device__ float operator()(const float &x) const { return exp(x); }
};

struct func_entropy_kernel {
    __host__ __device__ float operator()(const float &x) const { float val = x*log2(x); return (val != val) ? 0 : val; }
};

struct func_pow2 {
    __host__ __device__ float operator()(const float &x) const { return pow(2,x); }
};

__global__ void perplexity_search(float* __restrict__ sigmas, 
                                    float* __restrict__ lower_bound, 
                                    float* __restrict__ upper_bound,
                                    float* __restrict__ perplexity, 
                                    const float* __restrict__ pij, 
                                    const float target_perplexity, 
                                    const int N) 
{
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if (TID > N) return;

    // Compute the perplexity for this point
    float entropy = 0.0f; 
    for (int i = 0; i < N; i++) {
        if (i != TID) {
            entropy += pij[i + TID*N]*log2(pij[i + TID*N]);
        }
    }
    perplexity[TID] = pow(2,-1.0*entropy);

    if (perplexity[TID] > target_perplexity) {
        upper_bound[TID] = sigmas[TID];
        
    } else {
        lower_bound[TID] = sigmas[TID];
    }
    sigmas[TID] = (upper_bound[TID] + lower_bound[TID])/2.0f;
}

thrust::device_vector<float> compute_pij(hipblasHandle_t &handle, 
                                         thrust::device_vector<float> &points, 
                                         thrust::device_vector<float> &sigma, 
                                         const unsigned int N, 
                                         const unsigned int NDIMS) 
{
    thrust::device_vector<float> pij_vals(N * N);
    squared_pairwise_dist(handle, pij_vals, points, N, NDIMS);

    thrust::device_vector<float> sigma_squared(sigma.size());
    square(sigma, sigma_squared);
    
    broadcast_matrix_vector(pij_vals, sigma_squared, N, N, thrust::divides<float>(), 1, -2.0f);
    thrust::transform(pij_vals.begin(), pij_vals.end(), pij_vals.begin(), func_exp());
    zero_diagonal(pij_vals, N);
    // reduce_sum over rows
    auto sums = reduce_sum(handle, pij_vals, N, N, 1);
    // divide column by resulting vector
    broadcast_matrix_vector(pij_vals, sums, N, N, thrust::divides<float>(), 0, 1.0f);
    float alpha = 0.5f/N;
    float beta = 0.5f/N;
    thrust::device_vector<float> pij_output(N*N);
    cublasSafeCall(hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, &alpha, thrust::raw_pointer_cast(pij_vals.data()), N, 
                               &beta, thrust::raw_pointer_cast(pij_vals.data()), N, thrust::raw_pointer_cast(pij_output.data()), N));
    return pij_output;
}

/**
  * Gradient formula from http://www.jmlr.org/papers/volume9/vandermaaten08a/vandermaaten08a.pdf
  * 
  * Given by ->
  *     forces_i = 4 * \sum_j (pij - qij)(yi - yj)(1 + ||y_i - y_j||^2)^-1
  * 
  * Notation below - in comments, actual variables in the code are referred to by <varname>_ to differentiate from the mathematical quantities
  *                     It's hard to name variables correctly because we don't want to keep allocating more memory. There's probably a better solution than this though.
  */
float compute_gradients(hipblasHandle_t &handle, 
                        thrust::device_vector<float> &forces,
                        thrust::device_vector<float> &dist, 
                        thrust::device_vector<float> &ys, 
                        thrust::device_vector<float> &pij, 
                        thrust::device_vector<float> &qij, 
                        const unsigned int N,
                        float eta) 
{
    // dist_ = ||y_i - y_j||^2
    squared_pairwise_dist(handle, dist, ys, N, PROJDIM);
    // dist_ = (1 + ||y_i - y_j||^2)^-1
    thrust::transform(dist.begin(), dist.end(), dist.begin(), func_inc_inv());
    zero_diagonal(dist, N);

    // qij_ = (1 + ||y_i - y_j||^2)^-1 / \Sum_{k != i} (1 + ||y_i - y_k||^2)^-1
    thrust::copy(dist.begin(), dist.end(), qij.begin());
    auto sums = reduce_sum(handle, qij, N, N, 1);
    broadcast_matrix_vector(qij, sums, N, N, thrust::divides<float>(), 0, 1.0f);
    // Compute loss = \sum_ij pij * log(pij / qij)
    thrust::device_vector<float> loss_(N * N);
    thrust::transform(pij.begin(), pij.end(), qij.begin(), loss_.begin(), func_kl());
    zero_diagonal(loss_, N);

    // printarray(loss_, N, N);
    float loss = thrust::reduce(loss_.begin(), loss_.end(), 0.0f, thrust::plus<float>());

    // qij_ = pij - qij
    thrust::transform(pij.begin(), pij.end(), qij.begin(), qij.begin(), thrust::minus<float>());
    // qij_ = (pij - qij)(1 + ||y_i - y_j||^2)^-1
    thrust::transform(qij.begin(), qij.end(), dist.begin(), qij.begin(), thrust::multiplies<float>());

    // forces_ = \sum_j (pij - qij)(1 + ||y_i - y_j||^2)^-1
    float alpha = 1.0f;
    float beta = 0.0f;
    thrust::device_vector<float> ones(PROJDIM * N, 1.0f);
    cublasSafeCall(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, PROJDIM, N, &alpha, 
                                thrust::raw_pointer_cast(qij.data()), N, thrust::raw_pointer_cast(ones.data()), N, &beta, 
                                thrust::raw_pointer_cast(forces.data()), N));

    // forces_ = y_i * \sum_j (pij - qij)(1 + ||y_i - y_j||^2)^-1
    thrust::transform(forces.begin(), forces.end(), ys.begin(), forces.begin(), thrust::multiplies<float>());
    alpha = -4.0f * eta;
    beta = 4.0f * eta;
    // forces_ = 4 * y_i * \sum_j (pij - qij)(1 + ||y_i - y_j||^2)^-1 - 4 * \sum_j y_j(pij - qij)(1 + ||y_i - y_j||^2)^-1
    cublasSafeCall(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, PROJDIM, N, &alpha, 
                                thrust::raw_pointer_cast(qij.data()), N, thrust::raw_pointer_cast(ys.data()), N, &beta, 
                                thrust::raw_pointer_cast(forces.data()), N));

    return loss;
}

thrust::device_vector<float> naive_tsne(hipblasHandle_t &handle, 
                                        thrust::device_vector<float> &points, 
                                        const unsigned int N, 
                                        const unsigned int NDIMS)
{
    max_norm(points);

    // Choose the right sigmas
    std::cout << "Selecting sigmas to match perplexity..." << std::endl;
    float perplexity_target = 30.0f;
    float perplexity_diff = 1;

    thrust::device_vector<float> sigmas = rand_in_range(N, 0.0f, N/2.0f);
    thrust::device_vector<float> perplexity(N);
    thrust::device_vector<float> lbs(N, 0.0f);
    thrust::device_vector<float> ubs(N, 500.0*N);

    auto pij = compute_pij(handle, points, sigmas, N, NDIMS);
    int iters = 1;
    while (perplexity_diff > 1e-4 && iters < 500) {
        
        dim3 dimBlock(128);
        dim3 dimGrid(iDivUp(N, 128));
        perplexity_search<<<dimGrid, dimBlock>>>(thrust::raw_pointer_cast(sigmas.data()), 
                                                     thrust::raw_pointer_cast(lbs.data()), 
                                                     thrust::raw_pointer_cast(ubs.data()), 
                                                     thrust::raw_pointer_cast(perplexity.data()),
                                                     thrust::raw_pointer_cast(pij.data()), 
                                                     perplexity_target,
                                                     N);

        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );

        
        // printarray(sigmas, 1, N);
        // printarray(lbs, 1, N);
        // printarray(ubs, 1, N);
        // printarray(perplexity, 1, N);

        float perplexity_diff = abs(thrust::reduce(perplexity.begin(), perplexity.end())/((float) N) - perplexity_target);
        printf("Current perplexity delta after %d iterations: %0.5f\n", iters, perplexity_diff);

        pij = compute_pij(handle, points, sigmas, N, NDIMS);
        iters++;
    } 

    pij = compute_pij(handle, points, sigmas, N, NDIMS);
    
    thrust::device_vector<float> forces(N * PROJDIM);
    thrust::device_vector<float> ys = random_vector(N * PROJDIM);
    
    // Momentum variables
    thrust::device_vector<float> yt_1(N * PROJDIM);
    thrust::device_vector<float> momentum(N * PROJDIM);
    float momentum_weight = 0.9f;


    //printarray(ys, N, 2);
    thrust::device_vector<float> qij(N * N);
    thrust::device_vector<float> dist(N * N);
    float eta = 0.10f;
    float loss = 0.0f;//, prevloss = std::numeric_limits<float>::infinity();

    // Create a dump file for the points
    std::ofstream dump_file;
    dump_file.open ("dump.txt");
    float host_ys[N * PROJDIM];
    dump_file << N << " " << PROJDIM << std::endl;

    for (int i = 0; i < 1000; i++) {
        loss = compute_gradients(handle, forces, dist, ys, pij, qij, N, eta);
        

        // Compute the momentum
        thrust::transform(ys.begin(), ys.end(), yt_1.begin(), momentum.begin(), thrust::minus<float>());
        thrust::transform(momentum.begin(), momentum.end(), thrust::make_constant_iterator(momentum_weight), momentum.begin(), thrust::multiplies<float>() );
        thrust::copy(ys.begin(), ys.end(), yt_1.begin());

        // Apply the forces
        thrust::transform(ys.begin(), ys.end(), forces.begin(), ys.begin(), thrust::plus<float>());
        thrust::transform(ys.begin(), ys.end(), momentum.begin(), ys.begin(), thrust::plus<float>());
        
        // if (loss > prevloss)
            // eta /= 2.;
        if (i % 10 == 0)
            std::cout << "Iteration: " << i << ", Loss: " << loss << ", ForceMag: " << norm(forces) << std::endl;
        // prevloss = loss;

        // Dump the points
        thrust::copy(ys.begin(), ys.end(), host_ys);
        for (int i = 0; i < N; i++) {
            for (int j = 0; j < PROJDIM; j++) {
                dump_file << host_ys[i + j*N] << " ";
            }
            dump_file << std::endl;
        }
    }
    dump_file.close();
    return ys;
}

