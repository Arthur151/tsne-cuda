#include "hip/hip_runtime.h"
/**
 * @brief Implementation of naive T-SNE
 * 
 * @file naive_tsne.cu
 * @author David Chan
 * @date 2018-04-04
 */

 #include "naive_tsne.h"

struct func_inc_inv {
    __host__ __device__ float operator()(const float &x) const { return 1 / (x + 1); }
};

struct func_kl {
    __host__ __device__ float operator()(const float &x, const float &y) const { 
        return x == 0.0f ? 0.0f : x * (log(x) - log(y));
    }
};

struct func_exp {
    __host__ __device__ float operator()(const float &x) const { return exp(x); }
};

thrust::device_vector<float> compute_pij(hipblasHandle_t &handle, 
                                         thrust::device_vector<float> &points, 
                                         thrust::device_vector<float> &sigma, 
                                         const unsigned int N, 
                                         const unsigned int NDIMS) 
{
    thrust::device_vector<float> pij_vals(N * N);
    squared_pairwise_dist(handle, pij_vals, points, N, NDIMS);

    thrust::device_vector<float> sigma_squared(sigma.size());
    square(sigma, sigma_squared);
    
    broadcast_matrix_vector(pij_vals, sigma_squared, N, N, thrust::divides<float>(), 1, -2.0f);
    thrust::transform(pij_vals.begin(), pij_vals.end(), pij_vals.begin(), func_exp());
    zero_diagonal(pij_vals, N);
    // reduce_sum over rows
    auto sums = reduce_sum(handle, pij_vals, N, N, 1);
    // divide column by resulting vector
    broadcast_matrix_vector(pij_vals, sums, N, N, thrust::divides<float>(), 0, 1.0f);
    float alpha = 0.5f/N;
    float beta = 0.5f/N;
    thrust::device_vector<float> pij_output(N*N);
    cublasSafeCall(hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, &alpha, thrust::raw_pointer_cast(pij_vals.data()), N, 
                               &beta, thrust::raw_pointer_cast(pij_vals.data()), N, thrust::raw_pointer_cast(pij_output.data()), N));
    return pij_output;
}

/**
  * Gradient formula from http://www.jmlr.org/papers/volume9/vandermaaten08a/vandermaaten08a.pdf
  * 
  * Given by ->
  *     forces_i = 4 * \sum_j (pij - qij)(yi - yj)(1 + ||y_i - y_j||^2)^-1
  * 
  * Notation below - in comments, actual variables in the code are referred to by <varname>_ to differentiate from the mathematical quantities
  *                     It's hard to name variables correctly because we don't want to keep allocating more memory. There's probably a better solution than this though.
  */
float compute_gradients(hipblasHandle_t &handle, 
                        thrust::device_vector<float> &forces,
                        thrust::device_vector<float> &dist, 
                        thrust::device_vector<float> &ys, 
                        thrust::device_vector<float> &pij, 
                        thrust::device_vector<float> &qij, 
                        const unsigned int N,
                        float eta) 
{
    // dist_ = ||y_i - y_j||^2
    squared_pairwise_dist(handle, dist, ys, N, PROJDIM);
    // dist_ = (1 + ||y_i - y_j||^2)^-1
    thrust::transform(dist.begin(), dist.end(), dist.begin(), func_inc_inv());
    zero_diagonal(dist, N);

    // qij_ = (1 + ||y_i - y_j||^2)^-1 / \Sum_{k != i} (1 + ||y_i - y_k||^2)^-1
    thrust::copy(dist.begin(), dist.end(), qij.begin());
    auto sums = reduce_sum(handle, qij, N, N, 1);
    broadcast_matrix_vector(qij, sums, N, N, thrust::divides<float>(), 0, 1.0f);
    // Compute loss = \sum_ij pij * log(pij / qij)
    thrust::device_vector<float> loss_(N * N);
    thrust::transform(pij.begin(), pij.end(), qij.begin(), loss_.begin(), func_kl());
    zero_diagonal(loss_, N);

    // printarray(loss_, N, N);
    float loss = thrust::reduce(loss_.begin(), loss_.end(), 0.0f, thrust::plus<float>());

    // qij_ = pij - qij
    thrust::transform(pij.begin(), pij.end(), qij.begin(), qij.begin(), thrust::minus<float>());
    // qij_ = (pij - qij)(1 + ||y_i - y_j||^2)^-1
    thrust::transform(qij.begin(), qij.end(), dist.begin(), qij.begin(), thrust::multiplies<float>());

    // forces_ = \sum_j (pij - qij)(1 + ||y_i - y_j||^2)^-1
    float alpha = 1.0f;
    float beta = 0.0f;
    thrust::device_vector<float> ones(PROJDIM * N, 1.0f);
    cublasSafeCall(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, PROJDIM, N, &alpha, 
                                thrust::raw_pointer_cast(qij.data()), N, thrust::raw_pointer_cast(ones.data()), N, &beta, 
                                thrust::raw_pointer_cast(forces.data()), N));

    // forces_ = y_i * \sum_j (pij - qij)(1 + ||y_i - y_j||^2)^-1
    thrust::transform(forces.begin(), forces.end(), ys.begin(), forces.begin(), thrust::multiplies<float>());
    alpha = -4.0f * eta;
    beta = 4.0f * eta;
    // forces_ = 4 * y_i * \sum_j (pij - qij)(1 + ||y_i - y_j||^2)^-1 - 4 * \sum_j y_j(pij - qij)(1 + ||y_i - y_j||^2)^-1
    cublasSafeCall(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, PROJDIM, N, &alpha, 
                                thrust::raw_pointer_cast(qij.data()), N, thrust::raw_pointer_cast(ys.data()), N, &beta, 
                                thrust::raw_pointer_cast(forces.data()), N));

    return loss;
}

thrust::device_vector<float> naive_tsne(hipblasHandle_t &handle, 
                                        thrust::device_vector<float> &points, 
                                        const unsigned int N, 
                                        const unsigned int NDIMS)
{
    max_norm(points);
    thrust::device_vector<float> sigmas(N, 0.5f);
    auto pij = compute_pij(handle, points, sigmas, N, NDIMS);
    thrust::device_vector<float> forces(N * PROJDIM);
    thrust::device_vector<float> ys = random_vector(N * PROJDIM);
    printarray(ys, N, 2);
    thrust::device_vector<float> qij(N * N);
    thrust::device_vector<float> dist(N * N);
    float eta = 10.0f;
    float loss;//, prevloss = std::numeric_limits<float>::infinity();
    for (int i = 0; i < 1000; i++) {
        loss = compute_gradients(handle, forces, dist, ys, pij, qij, N, eta);
        thrust::transform(ys.begin(), ys.end(), forces.begin(), ys.begin(), thrust::plus<float>());
        // if (loss > prevloss)
            // eta /= 2.;
        if (i % 10 == 0)
            std::cout << "Iteration: " << i << ", Loss: " << loss << ", ForceMag: " << norm(forces) << std::endl;
        // prevloss = loss;
    }
    return ys;
}

