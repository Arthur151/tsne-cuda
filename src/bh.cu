#include "hip/hip_runtime.h"
/*
CUDA BarnesHut v3.1: Simulation of the gravitational forces
in a galactic cluster using the Barnes-Hut n-body algorithm

Copyright (c) 2013, Texas State University-San Marcos. All rights reserved.

Redistribution and use in source and binary forms, with or without modification,
are permitted for academic, research, experimental, or personal use provided that
the following conditions are met:

   * Redistributions of source code must retain the above copyright notice, 
     this list of conditions and the following disclaimer.
   * Redistributions in binary form must reproduce the above copyright notice,
     this list of conditions and the following disclaimer in the documentation
     and/or other materials provided with the distribution.
   * Neither the name of Texas State University-San Marcos nor the names of its
     contributors may be used to endorse or promote products derived from this
     software without specific prior written permission.

For all other uses, please contact the Office for Commercialization and Industry
Relations at Texas State University-San Marcos <http://www.txstate.edu/ocir/>.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED
IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED
OF THE POSSIBILITY OF SUCH DAMAGE.

Author: Martin Burtscher <burtscher@txstate.edu>
*/


#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "bh_tsne.h"

#ifdef __KEPLER__

// thread count
#define THREADS1 1024  /* must be a power of 2 */
#define THREADS2 1024
#define THREADS3 768
#define THREADS4 128
#define THREADS5 1024
#define THREADS6 1024

// block count = factor * #SMs
#define FACTOR1 2
#define FACTOR2 2
#define FACTOR3 1  /* must all be resident at the same time */
#define FACTOR4 4  /* must all be resident at the same time */
#define FACTOR5 2
#define FACTOR6 2

#else

// thread count
#define THREADS1 512  /* must be a power of 2 */
#define THREADS2 512
#define THREADS3 128
#define THREADS4 64
#define THREADS5 256
#define THREADS6 1024

// block count = factor * #SMs
#define FACTOR1 3
#define FACTOR2 3
#define FACTOR3 6  /* must all be resident at the same time */
#define FACTOR4 6  /* must all be resident at the same time */
#define FACTOR5 5
#define FACTOR6 1

#endif

#define WARPSIZE 32
#define MAXDEPTH 32

__device__ volatile int stepd, bottomd, maxdepthd;
__device__ unsigned int blkcntd;
__device__ volatile float radiusd;


/******************************************************************************/
/*** initialize memory ********************************************************/
/******************************************************************************/

__global__ void InitializationKernel(int * __restrict errd)
{
  *errd = 0;
  stepd = -1;
  maxdepthd = 1;
  blkcntd = 0;
}


/******************************************************************************/
/*** compute center and radius ************************************************/
/******************************************************************************/

__global__
__launch_bounds__(THREADS1, FACTOR1)
void BoundingBoxKernel(int nnodesd, 
                        int nbodiesd, 
                        volatile int * __restrict startd, 
                        volatile int * __restrict childd, 
                        volatile float * __restrict massd, 
                        volatile float * __restrict posxd, 
                        volatile float * __restrict posyd, 
                        volatile float * __restrict maxxd, 
                        volatile float * __restrict maxyd, 
                        volatile float * __restrict minxd, 
                        volatile float * __restrict minyd) 
{
  register int i, j, k, inc;
  register float val, minx, maxx, miny, maxy;
  __shared__ volatile float sminx[THREADS1], smaxx[THREADS1], sminy[THREADS1], smaxy[THREADS1];

  // initialize with valid data (in case #bodies < #threads)
  minx = maxx = posxd[0];
  miny = maxy = posyd[0];

  // scan all bodies
  i = threadIdx.x;
  inc = THREADS1 * gridDim.x;
  for (j = i + blockIdx.x * THREADS1; j < nbodiesd; j += inc) {
    val = posxd[j];
    minx = fminf(minx, val);
    maxx = fmaxf(maxx, val);
    val = posyd[j];
    miny = fminf(miny, val);
    maxy = fmaxf(maxy, val);
  }

  // reduction in shared memory
  sminx[i] = minx;
  smaxx[i] = maxx;
  sminy[i] = miny;
  smaxy[i] = maxy;

  for (j = THREADS1 / 2; j > 0; j /= 2) {
    __syncthreads();
    if (i < j) {
      k = i + j;
      sminx[i] = minx = fminf(minx, sminx[k]);
      smaxx[i] = maxx = fmaxf(maxx, smaxx[k]);
      sminy[i] = miny = fminf(miny, sminy[k]);
      smaxy[i] = maxy = fmaxf(maxy, smaxy[k]);
    }
  }

  // write block result to global memory
  if (i == 0) {
    k = blockIdx.x;
    minxd[k] = minx;
    maxxd[k] = maxx;
    minyd[k] = miny;
    maxyd[k] = maxy;
    __threadfence();

    inc = gridDim.x - 1;
    if (inc == atomicInc(&blkcntd, inc)) {
      // I'm the last block, so combine all block results
      for (j = 0; j <= inc; j++) {
        minx = fminf(minx, minxd[j]);
        maxx = fmaxf(maxx, maxxd[j]);
        miny = fminf(miny, minyd[j]);
        maxy = fmaxf(maxy, maxyd[j]);
      }

      // compute 'radius'
      radiusd = fmaxf(maxx - minx, maxy - miny) * 0.5f;

      // create root node
      k = nnodesd;
      bottomd = k;

      massd[k] = -1.0f;
      startd[k] = 0;
      posxd[k] = (minx + maxx) * 0.5f;
      posyd[k] = (miny + maxy) * 0.5f;
      k *= 4;
      for (i = 0; i < 4; i++) childd[k + i] = -1;

      stepd++;
    }
  }
}


/******************************************************************************/
/*** build tree ***************************************************************/
/******************************************************************************/

__global__
__launch_bounds__(1024, 1)
void ClearKernel1(int nnodesd, int nbodiesd, volatile int * __restrict childd)
{
  register int k, inc, top, bottom;

  top = 4 * nnodesd;
  bottom = 4 * nbodiesd;
  inc = blockDim.x * gridDim.x;
  k = (bottom & (-WARPSIZE)) + threadIdx.x + blockIdx.x * blockDim.x;  // align to warp size
  if (k < bottom) k += inc;

  // iterate over all cells assigned to thread
  while (k < top) {
    childd[k] = -1;
    k += inc;
  }
}


__global__
__launch_bounds__(THREADS2, FACTOR2)
void TreeBuildingKernel(int nnodesd, 
                        int nbodiesd, 
                        volatile int * __restrict errd, 
                        volatile int * __restrict childd, 
                        volatile float * __restrict posxd, 
                        volatile float * __restrict posyd) 
{
  register int i, j, depth, localmaxdepth, skip, inc;
  register float x, y, r;
  register float px, py;
  register float dx, dy;
  register int ch, n, cell, locked, patch;
  register float radius, rootx, rooty;

  // cache root data
  radius = radiusd;
  rootx = posxd[nnodesd];
  rooty = posyd[nnodesd];

  localmaxdepth = 1;
  skip = 1;
  inc = blockDim.x * gridDim.x;
  i = threadIdx.x + blockIdx.x * blockDim.x;

  // iterate over all bodies assigned to thread
  while (i < nbodiesd) {
    //   if (TID == 0)
        // printf("\tStarting\n");
    if (skip != 0) {
      // new body, so start traversing at root
      skip = 0;
      px = posxd[i];
      py = posyd[i];
      n = nnodesd;
      depth = 1;
      r = radius * 0.5f;
      dx = dy = -r;
      j = 0;
      // determine which child to follow
      if (rootx < px) {j = 1; dx = r;}
      if (rooty < py) {j |= 2; dy = r;}
      x = rootx + dx;
      y = rooty + dy;
    }

    // follow path to leaf cell
    ch = childd[n*4+j];
    while (ch >= nbodiesd) {
      n = ch;
      depth++;
      r *= 0.5f;
      dx = dy = -r;
      j = 0;
      // determine which child to follow
      if (x < px) {j = 1; dx = r;}
      if (y < py) {j |= 2; dy = r;}
      x += dx;
      y += dy;
      ch = childd[n*4+j];
    }
    if (ch != -2) {  // skip if child pointer is locked and try again later
      locked = n*4+j;
      if (ch == -1) {
        if (-1 == atomicCAS((int *)&childd[locked], -1, i)) {  // if null, just insert the new body
          localmaxdepth = max(depth, localmaxdepth);
          i += inc;  // move on to next body
          skip = 1;
        }
      } else {  // there already is a body in this position
        if (ch == atomicCAS((int *)&childd[locked], ch, -2)) {  // try to lock
          patch = -1;
          // create new cell(s) and insert the old and new body
          do {
            depth++;

            cell = atomicSub((int *)&bottomd, 1) - 1;
            if (cell <= nbodiesd) {
              *errd = 1;
              bottomd = nnodesd;
            }

            if (patch != -1) {
              childd[n*4+j] = cell;
            }
            patch = max(patch, cell);

            j = 0;
            if (x < posxd[ch]) j = 1;
            if (y < posyd[ch]) j |= 2;
            childd[cell*4+j] = ch;

            n = cell;
            r *= 0.5f;
            dx = dy = -r;
            j = 0;
            if (x < px) {j = 1; dx = r;}
            if (y < py) {j |= 2; dy = r;}
            x += dx;
            y += dy;

            ch = childd[n*4+j];
            // repeat until the two bodies are different children
          } while (ch >= 0);
          childd[n*4+j] = i;

          localmaxdepth = max(depth, localmaxdepth);
          i += inc;  // move on to next body
          skip = 2;
        }
      }
    }
    __syncthreads();  // __threadfence();

    if (skip == 2) {
      childd[locked] = patch;
    }
  }
  // record maximum tree depth
  atomicMax((int *)&maxdepthd, localmaxdepth);
}


__global__
__launch_bounds__(1024, 1)
void ClearKernel2(int nnodesd, volatile int * __restrict startd, volatile float * __restrict massd)
{
  register int k, inc, bottom;

  bottom = bottomd;
  inc = blockDim.x * gridDim.x;
  k = (bottom & (-WARPSIZE)) + threadIdx.x + blockIdx.x * blockDim.x;  // align to warp size
  if (k < bottom) k += inc;

  // iterate over all cells assigned to thread
  while (k < nnodesd) {
    massd[k] = -1.0f;
    startd[k] = -1;
    k += inc;
  }
}


/******************************************************************************/
/*** compute center of mass ***************************************************/
/******************************************************************************/

__global__
__launch_bounds__(THREADS3, FACTOR3)
void SummarizationKernel(const int nnodesd, 
                            const int nbodiesd, 
                            volatile int * __restrict countd, 
                            const int * __restrict childd, 
                            volatile float * __restrict massd, 
                            volatile float * __restrict posxd, 
                            volatile float * __restrict posyd) 
{
  register int i, j, k, ch, inc, cnt, bottom, flag;
  register float m, cm, px, py;
  __shared__ int child[THREADS3 * 4];
  __shared__ float mass[THREADS3 * 4];

  bottom = bottomd;
  inc = blockDim.x * gridDim.x;
  k = (bottom & (-WARPSIZE)) + threadIdx.x + blockIdx.x * blockDim.x;  // align to warp size
  if (k < bottom) k += inc;

  register int restart = k;
  for (j = 0; j < 5; j++) {  // wait-free pre-passes
    // iterate over all cells assigned to thread
    while (k <= nnodesd) {
      if (massd[k] < 0.0f) {
        for (i = 0; i < 4; i++) {
          ch = childd[k*4+i];
          child[i*THREADS3+threadIdx.x] = ch;  // cache children
          if ((ch >= nbodiesd) && ((mass[i*THREADS3+threadIdx.x] = massd[ch]) < 0.0f)) {
            break;
          }
        }
        if (i == 4) {
          // all children are ready
          cm = 0.0f;
          px = 0.0f;
          py = 0.0f;
          cnt = 0;
          for (i = 0; i < 4; i++) {
            ch = child[i*THREADS3+threadIdx.x];
            if (ch >= 0) {
              if (ch >= nbodiesd) {  // count bodies (needed later)
                m = mass[i*THREADS3+threadIdx.x];
                cnt += countd[ch];
              } else {
                m = massd[ch];
                cnt++;
              }
              // add child's contribution
              cm += m;
              px += posxd[ch] * m;
              py += posyd[ch] * m;
            }
          }
          countd[k] = cnt;
          m = 1.0f / cm;
          posxd[k] = px * m;
          posyd[k] = py * m;
          __threadfence();  // make sure data are visible before setting mass
          massd[k] = cm;
        }
      }
      k += inc;  // move on to next cell
    }
    k = restart;
  }

  flag = 0;
  j = 0;
  // iterate over all cells assigned to thread
  while (k <= nnodesd) {
    if (massd[k] >= 0.0f) {
      k += inc;
    } else {
      if (j == 0) {
        j = 4;
        for (i = 0; i < 4; i++) {
          ch = childd[k*4+i];
          child[i*THREADS3+threadIdx.x] = ch;  // cache children
          if ((ch < nbodiesd) || ((mass[i*THREADS3+threadIdx.x] = massd[ch]) >= 0.0f)) {
            j--;
          }
        }
      } else {
        j = 4;
        for (i = 0; i < 4; i++) {
          ch = child[i*THREADS3+threadIdx.x];
          if ((ch < nbodiesd) || (mass[i*THREADS3+threadIdx.x] >= 0.0f) || ((mass[i*THREADS3+threadIdx.x] = massd[ch]) >= 0.0f)) {
            j--;
          }
        }
      }

      if (j == 0) {
        // all children are ready
        cm = 0.0f;
        px = 0.0f;
        py = 0.0f;
        cnt = 0;
        for (i = 0; i < 4; i++) {
          ch = child[i*THREADS3+threadIdx.x];
          if (ch >= 0) {
            if (ch >= nbodiesd) {  // count bodies (needed later)
              m = mass[i*THREADS3+threadIdx.x];
              cnt += countd[ch];
            } else {
              m = massd[ch];
              cnt++;
            }
            // add child's contribution
            cm += m;
            px += posxd[ch] * m;
            py += posyd[ch] * m;
          }
        }
        countd[k] = cnt;
        m = 1.0f / cm;
        posxd[k] = px * m;
        posyd[k] = py * m;
        flag = 1;
      }
    }
    __syncthreads();  
    __threadfence();
    if (flag != 0) {
      massd[k] = cm;
      k += inc;
      flag = 0;
    }
  }
}


/******************************************************************************/
/*** sort bodies **************************************************************/
/******************************************************************************/

__global__
__launch_bounds__(THREADS4, FACTOR4)
void SortKernel(int nnodesd, int nbodiesd, int * __restrict sortd, int * __restrict countd, volatile int * __restrict startd, int * __restrict childd)
{
  register int i, j, k, ch, dec, start, bottom;

  bottom = bottomd;
  dec = blockDim.x * gridDim.x;
  k = nnodesd + 1 - dec + threadIdx.x + blockIdx.x * blockDim.x;

  // iterate over all cells assigned to thread
  while (k >= bottom) {
    start = startd[k];
    if (start >= 0) {
      j = 0;
      for (i = 0; i < 4; i++) {
        ch = childd[k*4+i];
        if (ch >= 0) {
          if (i != j) {
            // move children to front (needed later for speed)
            childd[k*4+i] = -1;
            childd[k*4+j] = ch;
          }
          j++;
          if (ch >= nbodiesd) {
            // child is a cell
            startd[ch] = start;  // set start ID of child
            start += countd[ch];  // add #bodies in subtree
          } else {
            // child is a body
            sortd[start] = ch;  // record body in 'sorted' array
            start++;
          }
        }
      }
      k -= dec;  // move on to next cell
    }
  }
}


/******************************************************************************/
/*** compute force ************************************************************/
/******************************************************************************/

__global__
__launch_bounds__(THREADS5, FACTOR5)
void ForceCalculationKernel(int nnodesd, 
                            int nbodiesd, 
                            volatile int * __restrict errd, 
                            float theta, 
                            volatile int * __restrict sortd, 
                            volatile int * __restrict childd, 
                            volatile float * __restrict massd, 
                            volatile float * __restrict posxd, 
                            volatile float * __restrict posyd, 
                            volatile float * __restrict velxd, 
                            volatile float * __restrict velyd,
                            volatile float * __restrict normd) 
{
  register int i, j, k, n, depth, base, sbase, diff, pd, nd;
  register float px, py, vx, vy, dx, dy, normsum, tmp, mult;
  __shared__ volatile int pos[MAXDEPTH * THREADS5/WARPSIZE], node[MAXDEPTH * THREADS5/WARPSIZE];
  __shared__ float dq[MAXDEPTH * THREADS5/WARPSIZE];

  if (0 == threadIdx.x) {
    // tmp = radiusd * 2;
    // precompute values that depend only on tree level
    dq[0] = radiusd * theta; //tmp * tmp * itolsqd;
    for (i = 1; i < maxdepthd; i++) {
      dq[i] = dq[i - 1] * 0.5f; // radius is halved with every level of the tree
      // dq[i - 1] += epssqd;
    }
    // dq[i - 1] += epssqd;

    if (maxdepthd > MAXDEPTH) {
      *errd = maxdepthd;
    }
  }
  __syncthreads();

  if (maxdepthd <= MAXDEPTH) {
    // figure out first thread in each warp (lane 0)
    base = threadIdx.x / WARPSIZE;
    sbase = base * WARPSIZE;
    j = base * MAXDEPTH;

    diff = threadIdx.x - sbase;
    // make multiple copies to avoid index calculations later
    if (diff < MAXDEPTH) {
      dq[diff+j] = dq[diff];
    }
    __syncthreads();
    __threadfence_block();

    // iterate over all bodies assigned to thread
    for (k = threadIdx.x + blockIdx.x * blockDim.x; k < nbodiesd; k += blockDim.x * gridDim.x) {
      i = sortd[k];  // get permuted/sorted index
      // cache position info
      px = posxd[i];
      py = posyd[i];

      vx = 0.0f;
      vy = 0.0f;
      normsum = 0.0f;

      // initialize iteration stack, i.e., push root node onto stack
      depth = j;
      if (sbase == threadIdx.x) {
        pos[j] = 0;
        node[j] = nnodesd * 4;
      }

      do {
        // stack is not empty
        pd = pos[depth];
        nd = node[depth];
        while (pd < 4) {
          // node on top of stack has more children to process
          n = childd[nd + pd];  // load child pointer
          pd++;

          if (n >= 0) {
            dx = posxd[n] - px;
            dy = posyd[n] - py;
            tmp = dx*dx + dy*dy; // distance squared
            // tmp = dx*dx + (dy*dy + epssqd) (why softening?)
            if ((n < nbodiesd) || __all(tmp >= dq[depth])) {  // check if all threads agree that cell is far enough away (or is a body)
            //   tmp = rsqrtf(tmp);  // compute distance
              // from sptree.cpp
              tmp = 1 / (1 + tmp);
              mult = massd[n] * tmp;
              normsum += mult;
              mult *= tmp;
              vx += dx * mult;
              vy += dy * mult;
            } else {
              // push cell onto stack
              if (sbase == threadIdx.x) {  // maybe don't push and inc if last child
                pos[depth] = pd;
                node[depth] = nd;
              }
              depth++;
              pd = 0;
              nd = n * 4;
            }
          } else {
            pd = 4;  // early out because all remaining children are also zero
          }
        }
        depth--;  // done with this level
      } while (depth >= j);

      if (stepd >= 0) {
        // update velocity
        // TODO: This is probably wrongish and depends on what I do in the attractive force calculation
        velxd[i] += vx;
        velyd[i] += vy;
        normd[i] = normsum;
      }
    }
  }
}


/******************************************************************************/
/*** advance bodies ***********************************************************/
/******************************************************************************/
// Edited to add momentum, repulsive, attr forces, etc.
__global__
__launch_bounds__(THREADS6, FACTOR6)
void IntegrationKernel(int N,
                        int nnodes,
                        float eta,
                        float norm,
                        float momentum,
                        volatile float * __restrict pts, // (nnodes + 1) x 2
                        volatile float * __restrict attr_forces, // (N x 2)
                        volatile float * __restrict rep_forces, // (nnodes + 1) x 2
                        volatile float * __restrict old_forces) // (N x 2)
{
  register int i, inc;
  register float tmpx, tmpy;

  // iterate over all bodies assigned to thread
  // TODO: fix momentum at step 0
  inc = blockDim.x * gridDim.x;
  for (i = threadIdx.x + blockIdx.x * blockDim.x; i < N; i += inc) {
      tmpx = 4.0f * (attr_forces[i] + (rep_forces[i] / norm));
      tmpy = 4.0f * (attr_forces[i + N] + (rep_forces[nnodes + 1 + i] / norm));
      tmpx = momentum * tmpx + (1 - momentum) * old_forces[i];
      tmpy = momentum * tmpy + (1 - momentum) * old_forces[i + N];
      pts[i] -= eta * tmpx;
      pts[i + nnodes + 1] -= eta * tmpy;
      old_forces[i] = tmpx;
      old_forces[i + N] = tmpy;
   }
}


/******************************************************************************/
/*** compute attractive force *************************************************/
/******************************************************************************/
__global__
void computePijxQij(int N, 
                    int nnz, 
                    int nnodes,
                    volatile float * __restrict pij,
                    volatile int   * __restrict pijRowPtr,
                    volatile int   * __restrict pijColInd,
                    volatile float * __restrict forceProd,
                    volatile float * __restrict pts)
{
    register int TID, i, j, start, end;
    register float ix, iy, jx, jy, dx, dy, tmp;
    TID = threadIdx.x + blockIdx.x * blockDim.x;
    if (TID >= nnz) return;
    start = 0; end = N + 1;
    i = (N + 1) >> 1;
    while (end - start > 1) {
      j = pijRowPtr[i];
      end = (j <= TID) ? end : i;
      start = (j > TID) ? start : i;
      // if (j > TID)
          // end = i;
      // else
          // start = i;
      i = (start + end) >> 1;
    }
    // if (!(pijRowPtr[i] <= TID && pijRowPtr[i + 1] > TID))
        // printf("something's wrong!\n");
    
    j = pijColInd[TID - i];
    
    ix = pts[i]; iy = pts[nnodes + 1 + i];
    jx = pts[j]; jy = pts[nnodes + 1 + j];
    dx = ix - jx;
    dy = iy - jy;
    tmp = (1 + dx*dx + dy*dy);
    forceProd[TID] = pij[TID] / tmp;
}

// computes unnormalized attractive forces
void computeAttrForce(int N,
                        int nnz,
                        int nnodes,
                        hipsparseHandle_t &handle,
                        hipsparseMatDescr_t &descr,
                        thrust::device_vector<float> &sparsePij,
                        thrust::device_vector<int>   &pijRowPtr, // (N + 1)-D vector, should be constant L
                        thrust::device_vector<int>   &pijColInd, // NxL matrix (same shape as sparsePij)
                        thrust::device_vector<float> &forceProd, // NxL matrix
                        thrust::device_vector<float> &pts,       // (nnodes + 1) x 2 matrix
                        thrust::device_vector<float> &forces,    // N x 2 matrix
                        thrust::device_vector<float> &ones)      // N x 2 matrix of ones
{
    const int BLOCKSIZE = 128;
    const int NBLOCKS = iDivUp(nnz, BLOCKSIZE);
    computePijxQij<<<NBLOCKS, BLOCKSIZE>>>(N, nnz, nnodes,
                                            thrust::raw_pointer_cast(sparsePij.data()),
                                            thrust::raw_pointer_cast(pijRowPtr.data()),
                                            thrust::raw_pointer_cast(pijColInd.data()),
                                            thrust::raw_pointer_cast(forceProd.data()),
                                            thrust::raw_pointer_cast(pts.data()));
    gpuErrchk(hipDeviceSynchronize());
    // compute forces_i = sum_j pij*qij*normalization*yi
    float alpha = 1.0f;
    float beta = 0.0f;
    cusparseSafeCall(hipsparseScsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            N, 2, N, nnz, &alpha, descr,
                            thrust::raw_pointer_cast(forceProd.data()),
                            thrust::raw_pointer_cast(pijRowPtr.data()),
                            thrust::raw_pointer_cast(pijColInd.data()),
                            thrust::raw_pointer_cast(ones.data()),
                            N, &beta, thrust::raw_pointer_cast(forces.data()),
                            N));
    gpuErrchk(hipDeviceSynchronize());
    thrust::transform(forces.begin(), forces.begin() + N, pts.begin(), forces.begin(), thrust::multiplies<float>());
    thrust::transform(forces.begin() + N, forces.end(), pts.begin() + nnodes + 1, forces.begin() + N, thrust::multiplies<float>());

    // compute forces_i = forces_i - sum_j pij*qij*normalization*yj
    alpha = -1.0f;
    beta = 1.0f;
    cusparseSafeCall(hipsparseScsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            N, 2, N, nnz, &alpha, descr,
                            thrust::raw_pointer_cast(forceProd.data()),
                            thrust::raw_pointer_cast(pijRowPtr.data()),
                            thrust::raw_pointer_cast(pijColInd.data()),
                            thrust::raw_pointer_cast(pts.data()),
                            nnodes + 1, &beta, thrust::raw_pointer_cast(forces.data()),
                            N));
    gpuErrchk(hipDeviceSynchronize());
    

}

//TODO: Remove NDIMS argument
void compute_pij(hipblasHandle_t &handle, 
                    thrust::device_vector<float> &pij,  
                    const thrust::device_vector<float> &knn_distances, 
                    const thrust::device_vector<float> &sigma,
                    const unsigned int N, 
                    const unsigned int K,
                    const unsigned int NDIMS) 
{
    // Square the distances
    Math::square(knn_distances, pij);

    // Square the sigmas
    // TODO: This allocates memory (we may want to fix it....)
    thrust::device_vector<float> sigma_squared(sigma.size());
    Math::square(sigma, sigma_squared);

    // PIJ is KxN. :)
    Broadcast::broadcast_matrix_vector(pij, sigma_squared, K, N, thrust::divides<float>(), 1, -2.0f); // Divide by -2sigma
    thrust::transform(pij.begin(), pij.end(), pij.begin(), func_exp()); //Exponentiate
}

void normalize_pij(hipblasHandle_t &handle, 
                    thrust::device_vector<float> &pij,
                    const unsigned int N,
                    const unsigned int K) 
{
    // Reduce::reduce_sum over cols
    auto sums = Reduce::reduce_sum(handle, pij, K, N, 0);
    // divide column by resulting vector
    Broadcast::broadcast_matrix_vector(pij, sums, K, N, thrust::divides<float>(), 1, 1.0f);
}

// TODO: Add -1 notification here...
__global__ void postprocess_matrix(float* matrix, 
                                    long* long_indices,
                                    int* indices,
                                    unsigned int N_POINTS,
                                    unsigned int K) 
{
    register int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if (TID >= N_POINTS*K) return;

    // Set pij to 0 for each of the broken values
    if (matrix[TID] == 1.0f) matrix[TID] = 0.0f;
    indices[TID] = (int) long_indices[TID];
    return;
}

struct saxpy_functor : public thrust::binary_function<float,float,float>
{
    const float a;

    saxpy_functor(float _a) : a(_a) {}
    __host__ __device__
        float operator()(const float& x, const float& y) const { 
            return a * x + y;
        }
};

struct func_kl {
  __host__ __device__ float operator()(const float &x, const float &y) const { 
      return x == 0.0f ? 0.0f : x * (log(x) - log(y));
  }
};

struct func_entropy_kernel {
  __host__ __device__ float operator()(const float &x) const { float val = x*log2(x); return (val != val || isinf(val)) ? 0 : val; }
};

struct func_pow2 {
  __host__ __device__ float operator()(const float &x) const { return pow(2,x); }
};

__global__ void upper_lower_assign_bh(float * __restrict__ sigmas,
                                      float * __restrict__ lower_bound,
                                      float * __restrict__ upper_bound,
                                      const float * __restrict__ perplexity,
                                      const float target_perplexity,
                                      const unsigned int N)
{
  int TID = threadIdx.x + blockIdx.x * blockDim.x;
  if (TID > N) return;

  if (perplexity[TID] > target_perplexity)
    upper_bound[TID] = sigmas[TID];
  else
    lower_bound[TID] = sigmas[TID];
  sigmas[TID] = (upper_bound[TID] + lower_bound[TID])/2.0f;
}

void thrust_search_perplexity(hipblasHandle_t &handle,
                                thrust::device_vector<float> &sigmas,
                                thrust::device_vector<float> &lower_bound,
                                thrust::device_vector<float> &upper_bound,
                                thrust::device_vector<float> &perplexity,
                                const thrust::device_vector<float> &pij,
                                const float target_perplexity,
                                const unsigned int N,
                                const unsigned int K)
{
//   std::cout << "pij:" << std::endl;
//   printarray(pij, N, K);
//   std::cout << std::endl;
  thrust::device_vector<float> entropy_(pij.size());
  thrust::transform(pij.begin(), pij.end(), entropy_.begin(), func_entropy_kernel());

//   std::cout << "entropy:" << std::endl;
//   printarray(entropy_, N, K);
//   std::cout << std::endl;

  auto neg_entropy = Reduce::reduce_alpha(handle, entropy_, K, N, -1.0f, 0);

//   std::cout << "neg_entropy:" << std::endl;
//   printarray(neg_entropy, 1, N);
//   std::cout << std::endl;
  
  thrust::transform(neg_entropy.begin(), neg_entropy.end(), perplexity.begin(), func_pow2());
 
//   std::cout << "perplexity:" << std::endl;
//   printarray(perplexity, 1, N);
//   std::cout << std::endl;

  const unsigned int BLOCKSIZE = 128;
  const unsigned int NBLOCKS = iDivUp(N, BLOCKSIZE);
  upper_lower_assign_bh<<<NBLOCKS,BLOCKSIZE>>>(thrust::raw_pointer_cast(sigmas.data()),
                thrust::raw_pointer_cast(lower_bound.data()),
                thrust::raw_pointer_cast(upper_bound.data()),
                thrust::raw_pointer_cast(perplexity.data()),
                target_perplexity,
                N);
//   std::cout << "sigmas" << std::endl;
//   printarray(sigmas, 1, N);
//   std::cout << std::endl;

}

thrust::device_vector<float> search_perplexity(hipblasHandle_t &handle,
                                                  thrust::device_vector<float> &knn_distances,
                                                  const float perplexity_target,
                                                  const float eps,
                                                  const unsigned int N,
                                                  const unsigned int K) 
{
    thrust::device_vector<float> sigmas(N, 500.0f);
    thrust::device_vector<float> best_sigmas(N);
    thrust::device_vector<float> perplexity(N);
    thrust::device_vector<float> lbs(N, 0.0f);
    thrust::device_vector<float> ubs(N, 1000.0f);
    thrust::device_vector<float> pij(N*K);

    compute_pij(handle, pij, knn_distances, sigmas, N, K, 0);
    normalize_pij(handle, pij, N, K);
    float best_perplexity = 1000.0f;
    float perplexity_diff = 50.0f;
    int iters = 0;
    while (perplexity_diff > eps) {
        thrust_search_perplexity(handle, sigmas, lbs, ubs, perplexity, pij, perplexity_target, N, K);
        perplexity_diff = abs(thrust::reduce(perplexity.begin(), perplexity.end())/((float) N) - perplexity_target);
        if (perplexity_diff < best_perplexity){
            best_perplexity = perplexity_diff;
            printf("!! Best perplexity found in %d iterations: %0.5f\n", iters, perplexity_diff);
            thrust::copy(sigmas.begin(), sigmas.end(), best_sigmas.begin());
        }
        compute_pij(handle, pij, knn_distances, sigmas, N, K, 0);
        normalize_pij(handle, pij, N, K);
        iters++;
    } // Close perplexity search

    return pij;
}


thrust::device_vector<float> BHTSNE::tsne(hipblasHandle_t &dense_handle, 
                                          hipsparseHandle_t &sparse_handle,
                                            float* points, 
                                            unsigned int N_POINTS, 
                                            unsigned int N_DIMS, 
                                            unsigned int PROJDIM, 
                                            float perplexity, 
                                            float learning_rate, 
                                            float early_ex, 
                                            unsigned int n_iter, 
                                            unsigned int n_iter_np, 
                                            float min_g_norm)
{

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(BoundingBoxKernel), hipFuncCachePreferShared);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(TreeBuildingKernel), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(ClearKernel1), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(ClearKernel2), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(SummarizationKernel), hipFuncCachePreferShared);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(SortKernel), hipFuncCachePreferL1);
    #ifdef __KEPLER__
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(ForceCalculationKernel), hipFuncCachePreferEqual);
    #else
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(ForceCalculationKernel), hipFuncCachePreferL1);
    #endif
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(IntegrationKernel), hipFuncCachePreferL1);

    //TODO: Make this an argument
    const unsigned int K = 1023 < N_POINTS ? 1023 : N_POINTS - 1; 
    float *knn_distances = new float[N_POINTS*K];
    memset(knn_distances, 0, N_POINTS * K * sizeof(float));
    long *knn_indices = new long[N_POINTS*K]; // Allocate memory for the indices on the CPU

    // Compute the KNNs and distances
    Distance::knn(points, knn_indices, knn_distances, N_DIMS, N_POINTS, K);

    // Copy the distances to the GPU
    thrust::device_vector<float> d_knn_distances(N_POINTS*K);
    thrust::copy(knn_distances, knn_distances + N_POINTS*K, d_knn_distances.begin());

    // printarray<float>(d_knn_distances, N_POINTS, K);

    // Normalize the knn distances - this may not be necessary
    // TODO: Need to filter out zeros from distances
    // TODO: Some point is fucked up
    // TODO: nprobe / nlist issue? Check if there are -1s floating around...
    Math::max_norm(d_knn_distances); // Here, the extra 0s floating around won't matter

    // printarray<float>(d_knn_distances, N_POINTS, K);
    
    // Compute the pij of the KNN distribution

    //TODO: Make these arguments
    thrust::device_vector<float> d_pij = search_perplexity(dense_handle, d_knn_distances, perplexity, 1e-4, N_POINTS, K);

    // thrust::device_vector<float> sigmas(N_POINTS, 0.3);
    // thrust::device_vector<float> d_pij(N_POINTS*K);
    // compute_pij(dense_handle, d_pij, d_knn_distances, sigmas, N_POINTS, K, N_DIMS);

    // std::cout << std::endl;
    // printarray<float>(d_pij, N_POINTS, K);

    // Clean up the d_knn_distances matrix
    d_knn_distances.clear();
    d_knn_distances.shrink_to_fit();
    
    // Allocate memory for the indices
    thrust::device_vector<long> d_knn_indices_long(N_POINTS*K);
    thrust::device_vector<int> d_knn_indices(N_POINTS*K);
    thrust::copy(knn_indices, knn_indices + N_POINTS*K, d_knn_indices_long.begin());

    // std::cout << std::endl;
    // printarray<long>(d_knn_indices_long, N_POINTS, K);

    // Post-process the pij matrix-indives to remove zero elements/check for -1 elements
    const int NBLOCKS_PP = iDivUp(N_POINTS*K, 128);
    postprocess_matrix<<< NBLOCKS_PP, 128 >>>(thrust::raw_pointer_cast(d_pij.data()), 
                                              thrust::raw_pointer_cast(d_knn_indices_long.data()), 
                                              thrust::raw_pointer_cast(d_knn_indices.data()),  N_POINTS, K);
    hipDeviceSynchronize();

    // Clean up extra memory
    d_knn_indices_long.clear();
    d_knn_indices_long.shrink_to_fit();
    delete[] knn_distances;
    delete[] knn_indices;

    // Normalize the pij matrix, we do this after post-processing to avoid issues
    // in the distribution caused by exponentiation.
    // normalize_pij(dense_handle, d_pij, N_POINTS, K);

    // Construct some additional descriptors for sparse matrix multiplication (for the symmetrization)
    hipsparseMatDescr_t descr;
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

    // Symmetrize d_pij
    thrust::device_vector<float> sparsePij; // Device
    thrust::device_vector<int> pijRowPtr; // Device
    thrust::device_vector<int> pijColInd; // Device
    int sym_nnz;
    Sparse::sym_mat_gpu(d_pij, d_knn_indices, sparsePij, pijColInd, pijRowPtr, &sym_nnz, N_POINTS, K);

    // Clear some old memory
    d_knn_indices.clear();
    d_knn_indices.shrink_to_fit();
    d_pij.clear();
    d_pij.shrink_to_fit();  

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    if (deviceProp.warpSize != WARPSIZE) {
      fprintf(stderr, "Warp size must be %d\n", deviceProp.warpSize);
      exit(-1);
    }

    int blocks = deviceProp.multiProcessorCount;

    int nnodes = N_POINTS * 2;
    if (nnodes < 1024*blocks) nnodes = 1024*blocks;
    while ((nnodes & (WARPSIZE-1)) != 0) nnodes++;
    nnodes--;

    thrust::device_vector<float> forceProd(sparsePij.size());
    thrust::device_vector<float> pts = Random::random_vector((nnodes + 1) * 2); //TODO: Rename this function
    thrust::device_vector<float> rep_forces((nnodes + 1) * 2, 0);
    thrust::device_vector<float> attr_forces(N_POINTS * 2, 0);
    thrust::device_vector<float> old_forces(N_POINTS * 2, 0); // for momentum

    thrust::device_vector<int> errl(1);
    thrust::device_vector<int> startl(nnodes + 1);
    thrust::device_vector<int> childl((nnodes + 1) * 4);
    thrust::device_vector<float> massl(nnodes + 1, 1.0); // TODO: probably don't need massl
    thrust::device_vector<int> countl(nnodes + 1);
    thrust::device_vector<int> sortl(nnodes + 1);
    thrust::device_vector<float> norml(nnodes + 1);

    thrust::device_vector<float> maxxl(blocks * FACTOR1);
    thrust::device_vector<float> maxyl(blocks * FACTOR1);
    thrust::device_vector<float> minxl(blocks * FACTOR1);
    thrust::device_vector<float> minyl(blocks * FACTOR1);
    
    thrust::device_vector<float> ones(N_POINTS * 2, 1); // This is for reduce summing, etc.

    float eta = learning_rate * early_ex;
    float momentum = 0.8f;
    float norm;
    
    // These variables currently govern the tolerance (whether it recurses on a cell)
    float theta = 0.5f;
    InitializationKernel<<<1, 1>>>(thrust::raw_pointer_cast(errl.data()));
    gpuErrchk(hipDeviceSynchronize());
    
    std::ofstream dump_file;
    dump_file.open("dump_ys.txt");
    float host_ys[(nnodes + 1) * 2];
    dump_file << N_POINTS << " " << 2 << std::endl;
    
    for (int step = 0; step < n_iter; step++) {
        thrust::fill(rep_forces.begin(), rep_forces.end(), 0);
        
        // Repulsive force with Barnes Hut
        BoundingBoxKernel<<<blocks * FACTOR1, THREADS1>>>(nnodes, 
                                                          N_POINTS, 
                                                          thrust::raw_pointer_cast(startl.data()), 
                                                          thrust::raw_pointer_cast(childl.data()), 
                                                          thrust::raw_pointer_cast(massl.data()), 
                                                          thrust::raw_pointer_cast(pts.data()), 
                                                          thrust::raw_pointer_cast(pts.data() + nnodes + 1), 
                                                          thrust::raw_pointer_cast(maxxl.data()), 
                                                          thrust::raw_pointer_cast(maxyl.data()), 
                                                          thrust::raw_pointer_cast(minxl.data()), 
                                                          thrust::raw_pointer_cast(minyl.data()));

        gpuErrchk(hipDeviceSynchronize());

        ClearKernel1<<<blocks * 1, 1024>>>(nnodes, N_POINTS, thrust::raw_pointer_cast(childl.data()));
        TreeBuildingKernel<<<blocks * FACTOR2, THREADS2>>>(nnodes, N_POINTS, thrust::raw_pointer_cast(errl.data()), 
                                                                             thrust::raw_pointer_cast(childl.data()), 
                                                                             thrust::raw_pointer_cast(pts.data()), 
                                                                             thrust::raw_pointer_cast(pts.data() + nnodes + 1));
        ClearKernel2<<<blocks * 1, 1024>>>(nnodes, thrust::raw_pointer_cast(startl.data()), thrust::raw_pointer_cast(massl.data()));
        gpuErrchk(hipDeviceSynchronize());
        
        SummarizationKernel<<<blocks * FACTOR3, THREADS3>>>(nnodes, N_POINTS, thrust::raw_pointer_cast(countl.data()), 
                                                                                      thrust::raw_pointer_cast(childl.data()), 
                                                                                      thrust::raw_pointer_cast(massl.data()),
                                                                                      thrust::raw_pointer_cast(pts.data()),
                                                                                      thrust::raw_pointer_cast(pts.data() + nnodes + 1));
        gpuErrchk(hipDeviceSynchronize());
        
        SortKernel<<<blocks * FACTOR4, THREADS4>>>(nnodes, N_POINTS, thrust::raw_pointer_cast(sortl.data()), 
                                                                     thrust::raw_pointer_cast(countl.data()), 
                                                                     thrust::raw_pointer_cast(startl.data()), 
                                                                     thrust::raw_pointer_cast(childl.data()));
        gpuErrchk(hipDeviceSynchronize());
        
        ForceCalculationKernel<<<blocks * FACTOR5, THREADS5>>>(nnodes, N_POINTS, thrust::raw_pointer_cast(errl.data()), 
                                                                    theta,
                                                                    thrust::raw_pointer_cast(sortl.data()), 
                                                                    thrust::raw_pointer_cast(childl.data()), 
                                                                    thrust::raw_pointer_cast(massl.data()), 
                                                                    thrust::raw_pointer_cast(pts.data()),
                                                                    thrust::raw_pointer_cast(pts.data() + nnodes + 1),
                                                                    thrust::raw_pointer_cast(rep_forces.data()),
                                                                    thrust::raw_pointer_cast(rep_forces.data() + nnodes + 1),
                                                                    thrust::raw_pointer_cast(norml.data()));
        gpuErrchk(hipDeviceSynchronize());

        // compute attractive forces
        computeAttrForce(N_POINTS, sparsePij.size(), nnodes, sparse_handle, descr, sparsePij, pijRowPtr, pijColInd, forceProd, pts, attr_forces, ones);
        gpuErrchk(hipDeviceSynchronize());
        
        norm = thrust::reduce(norml.begin(), norml.begin() + N_POINTS, 0.0f, thrust::plus<float>());
        if (step % 10 == 0)
            std::cout << "Step: " << step << ", Norm: " << norm << std::endl;
      
        IntegrationKernel<<<blocks * FACTOR6, THREADS6>>>(N_POINTS, nnodes, eta, norm, momentum, 
                                                                    thrust::raw_pointer_cast(pts.data()),
                                                                    thrust::raw_pointer_cast(attr_forces.data()),
                                                                    thrust::raw_pointer_cast(rep_forces.data()),
                                                                    thrust::raw_pointer_cast(old_forces.data()));


        if (step == 250) {eta /= early_ex;}
        // std::cout << "ATTR FORCES:" << std::endl;
        // for (int i = 0; i < 128; i++) {
            // std::cout << attr_forces[i] << ", " << attr_forces[i + N_POINTS] << std::endl;
        // }
        // std::cout << std::endl;
        // std::cout << "REP FORCES:" << std::endl;
        // for (int i = 0; i < N_POINTS; i++) {
            // std::cout << rep_forces[i] / norm << ", " << rep_forces[i + nnodes + 1] / norm << std::endl;
        // }
        // Add resulting force vector to positions w/ normalization, mul by 4 and learning rate
        // thrust::transform(rep_forces.begin(), rep_forces.begin() + N_POINTS, attr_forces.begin(), attr_forces.begin(), saxpy_functor(1 / norm));
        // thrust::transform(rep_forces.begin() + nnodes + 1, rep_forces.begin() + nnodes + 1 + N_POINTS, attr_forces.begin() + N_POINTS, attr_forces.begin() + N_POINTS, saxpy_functor(1 / norm));

        // thrust::transform(attr_forces.begin(), attr_forces.begin() + N_POINTS, pts.begin(), pts.begin(), saxpy_functor(-eta * 4.0f));
        // thrust::transform(attr_forces.begin() + N_POINTS, attr_forces.end(), pts.begin() + nnodes + 1, pts.begin() + nnodes + 1, saxpy_functor(-eta * 4.0f));
        // for (int i = 0; i < N_POINTS; i++) {
            // std::cout << attr_forces[i] << ", " << attr_forces[i + N_POINTS] << std::endl;
        // }
        
        thrust::copy(pts.begin(), pts.end(), host_ys);
        for (int i = 0; i < N_POINTS; i++) {
            dump_file << host_ys[i] << " " << host_ys[i + nnodes + 1] << std::endl;
        }
        // exit(1);
        // Done (check progress, etc.)
        // if (step >= 20)
            // exit(1);
    }
    dump_file.close();
    std::cout << "Fin." << std::endl;

    return pts;
}

/******************************************************************************/

// static void CudaTest(const char *msg)
// {
//   hipError_t e;

//   hipDeviceSynchronize();
//   if (hipSuccess != (e = hipGetLastError())) {
//     fprintf(stderr, "%s: %d\n", msg, e);
//     fprintf(stderr, "%s\n", hipGetErrorString(e));
//     exit(-1);
//   }
// }


/******************************************************************************/

// random number generator

#define MULT 1103515245
#define ADD 12345
#define MASK 0x7FFFFFFF
#define TWOTO31 2147483648.0

// static int A = 1;
// static int B = 0;
// static int randx = 1;
// static int lastrand;


// static void drndset(int seed)
// {
//    A = 1;
//    B = 0;
//    randx = (A * seed + B) & MASK;
//    A = (MULT * A) & MASK;
//    B = (MULT * B + ADD) & MASK;
// }


// static double drnd()
// {
//    lastrand = randx;
//    randx = (A * randx + B) & MASK;
//    return (double)lastrand / TWOTO31;
// }


/******************************************************************************/

// int main(int argc, char *argv[])
// {
//   register int i, run, blocks;
//   int nnodes, nbodies, step, timesteps;
//   register double runtime;
//   int error;
//   register float dtime, dthf, epssq, itolsq;
//   float time, timing[7];
//   hipEvent_t start, stop;
//   float *mass, *posx, *posy, *velx, *vely;

//   int *errl, *sortl, *childl, *countl, *startl;
//   float *massl;
//   float *posxl, *posyl;
//   float *velxl, *velyl;
//   float *maxxl, *maxyl;
//   float *minxl, *minyl;
//   float *norml;
//   register double rsc, vsc, r, v, x, y, sq, scale;

//   // perform some checks

//   printf("CUDA BarnesHut v3.1 ");
// #ifdef __KEPLER__
//   printf("[Kepler]\n");
// #else
//   printf("[Fermi]\n");
// #endif
//   printf("Copyright (c) 2013, Texas State University-San Marcos. All rights reserved.\n");
//   fflush(stdout);
//   if (argc != 4) {
//     fprintf(stderr, "\n");
//     fprintf(stderr, "arguments: number_of_bodies number_of_timesteps device\n");
//     exit(-1);
//   }

//   int deviceCount;
//   hipGetDeviceCount(&deviceCount);
//   if (deviceCount == 0) {
//     fprintf(stderr, "There is no device supporting CUDA\n");
//     exit(-1);
//   }

//   const int dev = atoi(argv[3]);
//   if ((dev < 0) || (deviceCount <= dev)) {
//     fprintf(stderr, "There is no device %d\n", dev);
//     exit(-1);
//   }
//   hipSetDevice(dev);

//   hipDeviceProp_t deviceProp;
//   hipGetDeviceProperties(&deviceProp, dev);
//   if ((deviceProp.major == 9999) && (deviceProp.minor == 9999)) {
//     fprintf(stderr, "There is no CUDA capable device\n");
//     exit(-1);
//   }
//   if (deviceProp.major < 2) {
//     fprintf(stderr, "Need at least compute capability 2.0\n");
//     exit(-1);
//   }
//   if (deviceProp.warpSize != WARPSIZE) {
//     fprintf(stderr, "Warp size must be %d\n", deviceProp.warpSize);
//     exit(-1);
//   }

//   blocks = deviceProp.multiProcessorCount;
// //  fprintf(stderr, "blocks = %d\n", blocks);

//   if ((WARPSIZE <= 0) || (WARPSIZE & (WARPSIZE-1) != 0)) {
//     fprintf(stderr, "Warp size must be greater than zero and a power of two\n");
//     exit(-1);
//   }
//   if (MAXDEPTH > WARPSIZE) {
//     fprintf(stderr, "MAXDEPTH must be less than or equal to WARPSIZE\n");
//     exit(-1);
//   }
//   if ((THREADS1 <= 0) || (THREADS1 & (THREADS1-1) != 0)) {
//     fprintf(stderr, "THREADS1 must be greater than zero and a power of two\n");
//     exit(-1);
//   }

//   // set L1/shared memory configuration
//   hipFuncSetCacheConfig(reinterpret_cast<const void*>(BoundingBoxKernel), hipFuncCachePreferShared);
//   hipFuncSetCacheConfig(reinterpret_cast<const void*>(TreeBuildingKernel), hipFuncCachePreferL1);
//   hipFuncSetCacheConfig(reinterpret_cast<const void*>(ClearKernel1), hipFuncCachePreferL1);
//   hipFuncSetCacheConfig(reinterpret_cast<const void*>(ClearKernel2), hipFuncCachePreferL1);
//   hipFuncSetCacheConfig(reinterpret_cast<const void*>(SummarizationKernel), hipFuncCachePreferShared);
//   hipFuncSetCacheConfig(reinterpret_cast<const void*>(SortKernel), hipFuncCachePreferL1);
// #ifdef __KEPLER__
//   hipFuncSetCacheConfig(reinterpret_cast<const void*>(ForceCalculationKernel), hipFuncCachePreferEqual);
// #else
//   hipFuncSetCacheConfig(reinterpret_cast<const void*>(ForceCalculationKernel), hipFuncCachePreferL1);
// #endif
//   hipFuncSetCacheConfig(reinterpret_cast<const void*>(IntegrationKernel), hipFuncCachePreferL1);

//   hipGetLastError();  // reset error value
//   for (run = 0; run < 3; run++) {
//     for (i = 0; i < 7; i++) timing[i] = 0.0f;

//     nbodies = atoi(argv[1]);
//     if (nbodies < 1) {
//       fprintf(stderr, "nbodies is too small: %d\n", nbodies);
//       exit(-1);
//     }
//     if (nbodies > (1 << 30)) {
//       fprintf(stderr, "nbodies is too large: %d\n", nbodies);
//       exit(-1);
//     }
//     nnodes = nbodies * 2;
//     if (nnodes < 1024*blocks) nnodes = 1024*blocks;
//     while ((nnodes & (WARPSIZE-1)) != 0) nnodes++;
//     nnodes--;

//     timesteps = atoi(argv[2]);
//     dtime = 0.025;  dthf = dtime * 0.5f;
//     epssq = 0.05 * 0.05;
//     itolsq = 1.0f / (0.5 * 0.5);

//     // allocate memory

//     if (run == 0) {
//       printf("configuration: %d bodies, %d time steps\n", nbodies, timesteps);

//       mass = (float *)malloc(sizeof(float) * nbodies);
//       if (mass == NULL) {fprintf(stderr, "cannot allocate mass\n");  exit(-1);}
//       posx = (float *)malloc(sizeof(float) * nbodies);
//       if (posx == NULL) {fprintf(stderr, "cannot allocate posx\n");  exit(-1);}
//       posy = (float *)malloc(sizeof(float) * nbodies);
//       if (posy == NULL) {fprintf(stderr, "cannot allocate posy\n");  exit(-1);}
//       velx = (float *)malloc(sizeof(float) * nbodies);
//       if (velx == NULL) {fprintf(stderr, "cannot allocate velx\n");  exit(-1);}
//       vely = (float *)malloc(sizeof(float) * nbodies);
//       if (vely == NULL) {fprintf(stderr, "cannot allocate vely\n");  exit(-1);}

//       if (hipSuccess != hipMalloc((void **)&errl, sizeof(int))) fprintf(stderr, "could not allocate errd\n");  CudaTest("couldn't allocate errd");
//       if (hipSuccess != hipMalloc((void **)&childl, sizeof(int) * (nnodes+1) * 4)) fprintf(stderr, "could not allocate childd\n");  CudaTest("couldn't allocate childd");
//       if (hipSuccess != hipMalloc((void **)&massl, sizeof(float) * (nnodes+1))) fprintf(stderr, "could not allocate massd\n");  CudaTest("couldn't allocate massd");
//       if (hipSuccess != hipMalloc((void **)&posxl, sizeof(float) * (nnodes+1))) fprintf(stderr, "could not allocate posxd\n");  CudaTest("couldn't allocate posxd");
//       if (hipSuccess != hipMalloc((void **)&posyl, sizeof(float) * (nnodes+1))) fprintf(stderr, "could not allocate posyd\n");  CudaTest("couldn't allocate posyd");
//       if (hipSuccess != hipMalloc((void **)&velxl, sizeof(float) * (nnodes+1))) fprintf(stderr, "could not allocate velxd\n");  CudaTest("couldn't allocate velxd");
//       if (hipSuccess != hipMalloc((void **)&velyl, sizeof(float) * (nnodes+1))) fprintf(stderr, "could not allocate velyd\n");  CudaTest("couldn't allocate velyd");
//       if (hipSuccess != hipMalloc((void **)&countl, sizeof(int) * (nnodes+1))) fprintf(stderr, "could not allocate countd\n");  CudaTest("couldn't allocate countd");
//       if (hipSuccess != hipMalloc((void **)&startl, sizeof(int) * (nnodes+1))) fprintf(stderr, "could not allocate startd\n");  CudaTest("couldn't allocate startd");
//       if (hipSuccess != hipMalloc((void **)&sortl, sizeof(int) * (nnodes+1))) fprintf(stderr, "could not allocate sortd\n");  CudaTest("couldn't allocate sortd");
//       if (hipSuccess != hipMalloc((void **)&norml, sizeof(int) * (nnodes+1))) fprintf(stderr, "could not allocate normd\n");  CudaTest("couldn't allocate normd");

//       if (hipSuccess != hipMalloc((void **)&maxxl, sizeof(float) * blocks * FACTOR1)) fprintf(stderr, "could not allocate maxxd\n");  CudaTest("couldn't allocate maxxd");
//       if (hipSuccess != hipMalloc((void **)&maxyl, sizeof(float) * blocks * FACTOR1)) fprintf(stderr, "could not allocate maxyd\n");  CudaTest("couldn't allocate maxyd");
//       if (hipSuccess != hipMalloc((void **)&minxl, sizeof(float) * blocks * FACTOR1)) fprintf(stderr, "could not allocate minxd\n");  CudaTest("couldn't allocate minxd");
//       if (hipSuccess != hipMalloc((void **)&minyl, sizeof(float) * blocks * FACTOR1)) fprintf(stderr, "could not allocate minyd\n");  CudaTest("couldn't allocate minyd");
//     }

//     // generate input

//     drndset(7);
//     rsc = (3 * 3.1415926535897932384626433832795) / 16;
//     vsc = sqrt(1.0 / rsc);
//     for (i = 0; i < nbodies; i++) {
//       mass[i] = 1.0 / nbodies;
//       r = 1.0 / sqrt(pow(drnd()*0.999, -2.0/3.0) - 1);
//       do {
//         x = drnd()*2.0 - 1.0;
//         y = drnd()*2.0 - 1.0;
//         sq = x*x + y*y;
//       } while (sq > 1.0);
//       scale = rsc * r / sqrt(sq);
//       posx[i] = x * scale;
//       posy[i] = y * scale;

//       do {
//         x = drnd();
//         y = drnd() * 0.1;
//       } while (y > x*x * pow(1 - x*x, 3.5));
//       v = x * sqrt(2.0 / sqrt(1 + r*r));
//       do {
//         x = drnd()*2.0 - 1.0;
//         y = drnd()*2.0 - 1.0;
//         sq = x*x + y*y;
//       } while (sq > 1.0);
//       scale = vsc * v / sqrt(sq);
//       velx[i] = x * scale;
//       vely[i] = y * scale;
//     }

//     if (hipSuccess != hipMemcpy(massl, mass, sizeof(float) * nbodies, hipMemcpyHostToDevice)) fprintf(stderr, "copying of mass to device failed\n");  CudaTest("mass copy to device failed");
//     if (hipSuccess != hipMemcpy(posxl, posx, sizeof(float) * nbodies, hipMemcpyHostToDevice)) fprintf(stderr, "copying of posx to device failed\n");  CudaTest("posx copy to device failed");
//     if (hipSuccess != hipMemcpy(posyl, posy, sizeof(float) * nbodies, hipMemcpyHostToDevice)) fprintf(stderr, "copying of posy to device failed\n");  CudaTest("posy copy to device failed");
//     if (hipSuccess != hipMemcpy(velxl, velx, sizeof(float) * nbodies, hipMemcpyHostToDevice)) fprintf(stderr, "copying of velx to device failed\n");  CudaTest("velx copy to device failed");
//     if (hipSuccess != hipMemcpy(velyl, vely, sizeof(float) * nbodies, hipMemcpyHostToDevice)) fprintf(stderr, "copying of vely to device failed\n");  CudaTest("vely copy to device failed");

//     // run timesteps (launch GPU kernels)

//     hipEventCreate(&start);  hipEventCreate(&stop);  
//     struct timeval starttime, endtime;
//     gettimeofday(&starttime, NULL);

//     hipEventRecord(start, 0);
//     InitializationKernel<<<1, 1>>>(errl);
//     hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
//     timing[0] += time;
//     CudaTest("kernel 0 launch failed");

//     for (step = 0; step < timesteps; step++) {
//       hipEventRecord(start, 0);
//       BoundingBoxKernel<<<blocks * FACTOR1, THREADS1>>>(nnodes, nbodies, startl, childl, massl, posxl, posyl, maxxl, maxyl, minxl, minyl);
//       hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
//       timing[1] += time;
//       CudaTest("kernel 1 launch failed");

//       hipEventRecord(start, 0);
//       ClearKernel1<<<blocks * 1, 1024>>>(nnodes, nbodies, childl);
//       TreeBuildingKernel<<<blocks * FACTOR2, THREADS2>>>(nnodes, nbodies, errl, childl, posxl, posyl);
//       ClearKernel2<<<blocks * 1, 1024>>>(nnodes, startl, massl);
//       hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
//       timing[2] += time;
//       CudaTest("kernel 2 launch failed");

//       hipEventRecord(start, 0);
//       SummarizationKernel<<<blocks * FACTOR3, THREADS3>>>(nnodes, nbodies, countl, childl, massl, posxl, posyl);
//       hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
//       timing[3] += time;
//       CudaTest("kernel 3 launch failed");

//       hipEventRecord(start, 0);
//       SortKernel<<<blocks * FACTOR4, THREADS4>>>(nnodes, nbodies, sortl, countl, startl, childl);
//       hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
//       timing[4] += time;
//       CudaTest("kernel 4 launch failed");

//       hipEventRecord(start, 0);
//       ForceCalculationKernel<<<blocks * FACTOR5, THREADS5>>>(nnodes, nbodies, errl, itolsq, epssq, sortl, childl, massl, posxl, posyl, velxl, velyl, norml);
//       hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
//       timing[5] += time;
//       CudaTest("kernel 5 launch failed");

//       hipEventRecord(start, 0);
//       IntegrationKernel<<<blocks * FACTOR6, THREADS6>>>(nbodies, dtime, posxl, posyl, velxl, velyl);
//       hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
//       timing[6] += time;
//       CudaTest("kernel 6 launch failed");
//     }
//     CudaTest("kernel launch failed");
//     hipEventDestroy(start);  hipEventDestroy(stop);

//     // transfer result back to CPU
//     if (hipSuccess != hipMemcpy(&error, errl, sizeof(int), hipMemcpyDeviceToHost)) fprintf(stderr, "copying of err from device failed\n");  CudaTest("err copy from device failed");
//     if (hipSuccess != hipMemcpy(posx, posxl, sizeof(float) * nbodies, hipMemcpyDeviceToHost)) fprintf(stderr, "copying of posx from device failed\n");  CudaTest("posx copy from device failed");
//     if (hipSuccess != hipMemcpy(posy, posyl, sizeof(float) * nbodies, hipMemcpyDeviceToHost)) fprintf(stderr, "copying of posy from device failed\n");  CudaTest("posy copy from device failed");
//     if (hipSuccess != hipMemcpy(velx, velxl, sizeof(float) * nbodies, hipMemcpyDeviceToHost)) fprintf(stderr, "copying of velx from device failed\n");  CudaTest("velx copy from device failed");
//     if (hipSuccess != hipMemcpy(vely, velyl, sizeof(float) * nbodies, hipMemcpyDeviceToHost)) fprintf(stderr, "copying of vely from device failed\n");  CudaTest("vely copy from device failed");

//     gettimeofday(&endtime, NULL);
//     runtime = endtime.tv_sec + endtime.tv_usec/1000000.0 - starttime.tv_sec - starttime.tv_usec/1000000.0;

//     printf("runtime: %.4lf s  (", runtime);
//     time = 0;
//     for (i = 1; i < 7; i++) {
//       printf(" %.1f ", timing[i]);
//       time += timing[i];
//     }
//     if (error == 0) {
//       printf(") = %.1f ms\n", time);
//     } else {
//       printf(") = %.1f ms FAILED %d\n", time, error);
//     }
//   }

//   // print output
//   i = 0;
// //  for (i = 0; i < nbodies; i++) {
//     printf("%.2e %.2e\n", posx[i], posy[i]);
// //  }

//   free(mass);
//   free(posx);
//   free(posy);
//   free(velx);
//   free(vely);

//   hipFree(errl);
//   hipFree(childl);
//   hipFree(massl);
//   hipFree(posxl);
//   hipFree(posyl);
//   hipFree(countl);
//   hipFree(startl);
//   hipFree(norml);

//   hipFree(maxxl);
//   hipFree(maxyl);
//   hipFree(minxl);
//   hipFree(minyl);

//   return 0;
// }
