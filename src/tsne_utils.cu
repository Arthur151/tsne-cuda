#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <float.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <random>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdexcept>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/sequence.h>

#include "tsne_utils.cuh"
#include "Utilities.cuh"

#define PROJDIM 2

struct func_square {
	__host__ __device__ double operator()(const float &x) const { return x * x; }
};

struct func_sqrt {
    __host__ __device__ double operator()(const float &x) const { return pow(x, 0.5); }
};

struct func_exp {
    __host__ __device__ double operator()(const float &x) const { return exp(x); }
};

struct func_inv {
    __host__ __device__ double operator()(const float &x) const { return pow(x, -1.0); }
};

struct func_inc {
    __host__ __device__ double operator()(const float &x) const { return x + 1; }
};

struct func_inc_inv {
    __host__ __device__ double operator()(const float &x) const { return pow(x + 1, -1.0); }
};

struct func_ln {
    __host__ __device__ double operator()(const float &x) const { return log(x); }
};

struct func_kl {
    __host__ __device__ double operator()(const float &x, const float &y) const { return y < 1e-4 ? 0 : x * log(x / y); }
};

struct prg
{
    float a, b;

    __host__ __device__
    prg(float _a=-1.f, float _b=1.f) : a(_a), b(_b) {};

    __host__ __device__
        float operator()(const unsigned int n) const
        {
            thrust::default_random_engine rng;
            thrust::uniform_real_distribution<float> dist(a, b);
            rng.discard(n);

            return dist(rng);
        }
};

__global__ void assemble_final_result(const float * __restrict__ d_norms_x_2, float * __restrict__ d_dots,
									  const int N) {

	const int i = threadIdx.x + blockIdx.x * gridDim.x;
	const int j = threadIdx.y + blockIdx.y * gridDim.y;

	if ((i < N) && (j < N)) d_dots[i * N + j] = d_norms_x_2[j] + d_norms_x_2[i] - 2 * d_dots[i * N + j];
    
}

// Performs the operation matrix[i,:] = matrix[i,:] - alpha*vector for each row 0 <= i < N
// This is in place addition
__global__ void _add_row_vec(float * __restrict__ matrix, const float * __restrict__ vector, const unsigned int N, const unsigned int M, const float alpha) {
    const unsigned int TID = threadIdx.x + blockIdx.x * gridDim.x;
    const unsigned int i = TID % N;
    const unsigned int j = TID / N;

    if (j < M) matrix[j * N + i] = matrix[j * N + i] + alpha*vector[j];
}

void add_row_vec(thrust::device_vector<float> &matrix, thrust::device_vector<float> &vector, const unsigned int N, const unsigned int M, const float alpha) {
    const unsigned int BLOCKSIZE = 32;
    const unsigned int NBLOCKS = iDivUp(N * M, BLOCKSIZE);
    _add_row_vec<<<NBLOCKS,BLOCKSIZE>>>(thrust::raw_pointer_cast(matrix.data()), thrust::raw_pointer_cast(vector.data()), N, M, alpha);
}

// Performs the operation matrix[i,:] = alpha*matrix[i,:]*vector for each row 0 <= i < N
__global__ void _mul_row_vec(float * __restrict__ matrix, const float * __restrict__ vector, const unsigned int N, const unsigned int M, const float alpha) {
    const unsigned int TID = threadIdx.x + blockIdx.x * gridDim.x;
    const unsigned int i = TID % N;
    const unsigned int j = TID / N;

    if (j < M) matrix[j * N + i] = alpha*matrix[j * N + i]*vector[j];
}

void mul_row_vec(thrust::device_vector<float> &matrix, thrust::device_vector<float> &vector, const unsigned int N, const unsigned int M, const float alpha) {
    const unsigned int BLOCKSIZE = 32;
    const unsigned int NBLOCKS = iDivUp(N * M, BLOCKSIZE);
    _mul_row_vec<<<NBLOCKS,BLOCKSIZE>>>(thrust::raw_pointer_cast(matrix.data()), thrust::raw_pointer_cast(vector.data()), N, M, alpha);
}

// Performs the operation matrix[i,:] = alpha*matrix[i,:]/vector for each row 0 <= i < N
__global__ void _div_row_vec(float * __restrict__ matrix, const float * __restrict__ vector, const unsigned int N, const unsigned int M, const float alpha) {
    const unsigned int TID = threadIdx.x + blockIdx.x * gridDim.x;
    const unsigned int i = TID % N;
    const unsigned int j = TID / N;

    if (j < M) 
        matrix[j * N + i] = alpha*matrix[j * N + i]/vector[j];
}

void div_row_vec(thrust::device_vector<float> &matrix, thrust::device_vector<float> &vector, const unsigned int N, const unsigned int M, const float alpha) {
    const unsigned int BLOCKSIZE = 32;
    const unsigned int NBLOCKS = iDivUp(N * M, BLOCKSIZE);
    _div_row_vec<<<NBLOCKS,BLOCKSIZE>>>(thrust::raw_pointer_cast(matrix.data()), thrust::raw_pointer_cast(vector.data()), N, M, alpha);
}

// Code from https://github.com/OrangeOwlSolutions/cuBLAS/blob/master/All_pairs_distances.cu
// Expects N x NDIMS matrix in points
void pairwise_dist(hipblasHandle_t &handle, thrust::device_vector<float> &distances, const thrust::device_vector<float> &points, const unsigned int N, const unsigned int NDIMS) {
    const unsigned int BLOCKSIZE = 16;

    auto squared_vals = square(points, N * NDIMS);
    auto squared_norms = reduce_sum(handle, squared_vals, N, NDIMS, 1);
    
    float alpha = 1.f;
    float beta = 0.f;
    // Could replace this with hipblasSsyrk, might be faster?
	cublasSafeCall(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, NDIMS, &alpha,
		                       thrust::raw_pointer_cast(points.data()), N, thrust::raw_pointer_cast(points.data()), N, &beta,
							   thrust::raw_pointer_cast(distances.data()), N));
 
	dim3 dimBlock(BLOCKSIZE, BLOCKSIZE);
	dim3 dimGrid(iDivUp(N, BLOCKSIZE), iDivUp(N, BLOCKSIZE));
	assemble_final_result<<<dimGrid, dimBlock>>>(thrust::raw_pointer_cast(squared_norms.data()), 
		                                         thrust::raw_pointer_cast(distances.data()), N);
}

void gauss_normalize(hipblasHandle_t &handle, thrust::device_vector<float> &points, const unsigned int N, const unsigned int NDIMS) {
    auto means = reduce_mean(handle, points, N, NDIMS, 0);

    // zero center
    add_row_vec(points, means, N, NDIMS, -1.f);
    
    // compute standard deviation
    auto squared_vals = square(points, N * NDIMS);
    auto norm_sum_of_squares = reduce_alpha(handle, squared_vals, N, NDIMS, 1.f / (N - 1), 0);
    auto stddev = sqrt(norm_sum_of_squares, N * NDIMS);

    // normalize
    div_row_vec(points, stddev, N, NDIMS, 1.f);
}

thrust::device_vector<float> square(const thrust::device_vector<float> &vec, const unsigned int N) {
    thrust::device_vector<float> squared_vals(N);
    thrust::transform(vec.begin(), vec.end(), squared_vals.begin(), func_square());
    return squared_vals;
}

thrust::device_vector<float> sqrt(const thrust::device_vector<float> &vec, const unsigned int N) {
    thrust::device_vector<float> sqrt_vals(N);
    thrust::transform(vec.begin(), vec.end(), sqrt_vals.begin(), func_sqrt());
    return sqrt_vals;
}

thrust::device_vector<float> compute_pij(hipblasHandle_t &handle, thrust::device_vector<float> &points, thrust::device_vector<float> &sigma, const unsigned int N, const unsigned int NDIMS) {
    thrust::device_vector<float> pij_vals(N * N);
    pairwise_dist(handle, pij_vals, points, N, NDIMS);
    auto sigma_squared = square(sigma, N);

    printf("pij Min: %0.5f \n", thrust::reduce(pij_vals.begin(), pij_vals.end(), 5000.0f, thrust::minimum<float>()));
    printf("pij Max: %0.5f \n", thrust::reduce(pij_vals.begin(), pij_vals.end(), 0.0f, thrust::maximum<float>()));

    // divide columns by -2*sigma_i^2
    div_row_vec(pij_vals, sigma_squared, N, N, -0.5f);

    printf("pij Min: %0.5f \n", thrust::reduce(pij_vals.begin(), pij_vals.end(), 5000.0f, thrust::minimum<float>()));
    printf("pij Max: %0.5f \n", thrust::reduce(pij_vals.begin(), pij_vals.end(), 0.0f, thrust::maximum<float>()));

    // exponentiate
    thrust::transform(pij_vals.begin(), pij_vals.end(), pij_vals.begin(), func_exp());
    // reduce_sum over rows (subtract one from result to deal with x_i == x_k)
    thrust::device_vector<float> ones(N, 1.f);
    thrust::device_vector<float> sums(N, -1.f);
    float alpha = 1.f;
    float beta = 1.f;
    cublasSafeCall(hipblasSgemv(handle, HIPBLAS_OP_N, N, N, &alpha, thrust::raw_pointer_cast(pij_vals.data()), N,
                                thrust::raw_pointer_cast(ones.data()), 1, &beta, thrust::raw_pointer_cast(sums.data()), 1));
    // divide column by resulting vector
    div_row_vec(pij_vals, sums, N, N, 1.0f);

    alpha = 0.5f/N;
    beta = 0.5f/N;
    thrust::device_vector<float> pij_output(N*N);
    cublasSafeCall(hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, &alpha, thrust::raw_pointer_cast(pij_vals.data()), N, 
                               &beta, thrust::raw_pointer_cast(pij_vals.data()), N, thrust::raw_pointer_cast(pij_output.data()), N));

    return pij_output;
}

float compute_gradients(hipblasHandle_t &handle, 
                        thrust::device_vector<float> &forces,
                        thrust::device_vector<float> &dist, 
                        thrust::device_vector<float> &ys, 
                        thrust::device_vector<float> &pij, 
                        thrust::device_vector<float> &qij, 
                        const unsigned int N,
                        float eta) 
{
    pairwise_dist(handle, dist, ys, N, PROJDIM);
    // dist = (1 + ||y_i - y_j||^2)^-1
    thrust::transform(dist.begin(), dist.end(), dist.begin(), func_inc_inv());

    thrust::device_vector<float> ones(N, 1.f);
    thrust::device_vector<float> sums(N, -1.f);
    float alpha = 1.f;
    float beta = 1.f;
    cublasSafeCall(hipblasSgemv(handle, HIPBLAS_OP_N, N, N, &alpha, thrust::raw_pointer_cast(dist.data()), N,
                                thrust::raw_pointer_cast(ones.data()), 1, &beta, thrust::raw_pointer_cast(sums.data()), 1));
    thrust::copy(dist.begin(), dist.end(), qij.begin());
    // qij = (1 + ||y_i - y_j||^2)^-1 / \Sum_{k != i} (1 + ||y_i - y_k||^2)^-1
    div_row_vec(qij, sums, N, N, 1.0f);

    thrust::device_vector<float> loss_(N * N);
    thrust::transform(pij.begin(), pij.end(), qij.begin(), loss_.begin(), func_kl());

    float loss = thrust::reduce(loss_.begin(), loss_.end(), 5.0f, thrust::minimum<float>());

    printf("dist Min: %0.5f \n", thrust::reduce(dist.begin(), dist.end(), 5000.0f, thrust::minimum<float>()));
    printf("dist Max: %0.5f \n", thrust::reduce(dist.begin(), dist.end(), 0.0f, thrust::maximum<float>()));

    printf("Qij Min: %0.5f \n", thrust::reduce(qij.begin(), qij.end(), 5000.0f, thrust::minimum<float>()));
    printf("Qij Max: %0.5f \n", thrust::reduce(qij.begin(), qij.end(), 0.0f, thrust::maximum<float>()));

    printf("pij Min: %0.5f \n", thrust::reduce(pij.begin(), pij.end(), 5000.0f, thrust::minimum<float>()));
    printf("pij Max: %0.5f \n", thrust::reduce(pij.begin(), pij.end(), 0.0f, thrust::maximum<float>()));

    // qij = pij - qij
    thrust::transform(pij.begin(), pij.end(), qij.begin(), qij.begin(), thrust::minus<float>());

    printf("Qij Min: %0.5f \n", thrust::reduce(qij.begin(), qij.end(), 5000.0f, thrust::minimum<float>()));
    printf("Qij Max: %0.5f \n", thrust::reduce(qij.begin(), qij.end(), 0.0f, thrust::maximum<float>()));
    
    // qij = (pij - qij) .* (1 + ||y_i - y_j||^2)^-1
    thrust::transform(qij.begin(), qij.end(), dist.begin(), qij.begin(), thrust::multiplies<float>());

    printf("Qij Min: %0.5f \n", thrust::reduce(qij.begin(), qij.end(), 5000.0f, thrust::minimum<float>()));
    printf("Qij Max: %0.5f \n", thrust::reduce(qij.begin(), qij.end(), 0.0f, thrust::maximum<float>()));

    cublasSafeCall(hipblasSgemv(handle, HIPBLAS_OP_N, N, N, &alpha, thrust::raw_pointer_cast(qij.data()), N,
                                thrust::raw_pointer_cast(ones.data()), 1, &beta, thrust::raw_pointer_cast(forces.data()), 1));

    printf("Qij Min: %0.5f \n", thrust::reduce(qij.begin(), qij.end(), 5000.0f, thrust::minimum<float>()));
    printf("Qij Max: %0.5f \n", thrust::reduce(qij.begin(), qij.end(), 0.0f, thrust::maximum<float>()));

    // TODO: needs to change for 3 dimensions
    thrust::copy(forces.begin(), forces.begin() + N, forces.begin() + N);

    // forces = A * ones(N, 1) .* ys
    thrust::transform(forces.begin(), forces.end(), ys.begin(), forces.begin(), thrust::multiplies<float>());

    printf("forces Min: %0.5f \n", thrust::reduce(forces.begin(), forces.end(), 5000.0f, thrust::minimum<float>()));
    printf("forces Max: %0.5f \n", thrust::reduce(forces.begin(), forces.end(), 0.0f, thrust::maximum<float>()));

    alpha = -4.0f * eta;
    beta = 4.0f * eta;
    // TODO: needs to change for 3 dimensions
    cublasSafeCall(hipblasSgemv(handle, HIPBLAS_OP_N, N, N, &alpha, thrust::raw_pointer_cast(qij.data()), N,
                                thrust::raw_pointer_cast(ys.data()), 1, &beta, thrust::raw_pointer_cast(forces.data()), 1));
    cublasSafeCall(hipblasSgemv(handle, HIPBLAS_OP_N, N, N, &alpha, thrust::raw_pointer_cast(qij.data()), N,
                                thrust::raw_pointer_cast(ys.data() + N), 1, &beta, thrust::raw_pointer_cast(forces.data() + N), 1));

    printf("forces Min: %0.5f \n", thrust::reduce(forces.begin(), forces.end(), 5000.0f, thrust::minimum<float>()));
    printf("forces Max: %0.5f \n", thrust::reduce(forces.begin(), forces.end(), 0.0f, thrust::maximum<float>()));

    
    return loss;
}

// expects matrix of size N x M
thrust::device_vector<float> reduce_alpha(hipblasHandle_t &handle, const thrust::device_vector<float> &matrix, const unsigned int N, const unsigned int M, float alpha, const int axis) {
    if (axis == 0) {
        thrust::device_vector<float> ones(N, 1.f);
        thrust::device_vector<float> means(M);

        float beta = 0.f;
        cublasSafeCall(hipblasSgemv(handle, HIPBLAS_OP_T, N, M, &alpha, thrust::raw_pointer_cast(matrix.data()), N,
                                    thrust::raw_pointer_cast(ones.data()), 1, &beta, thrust::raw_pointer_cast(means.data()), 1));
        return means;
    } else if (axis == 1) {
        thrust::device_vector<float> ones(M, 1.f);
        thrust::device_vector<float> means(N);

        float beta = 0.f;
        cublasSafeCall(hipblasSgemv(handle, HIPBLAS_OP_N, N, M, &alpha, thrust::raw_pointer_cast(matrix.data()), N,
                                    thrust::raw_pointer_cast(ones.data()), 1, &beta, thrust::raw_pointer_cast(means.data()), 1));
        return means;
    } else {
        throw std::runtime_error("Axis must be 0 or 1.");
    }
}

thrust::device_vector<float> reduce_mean(hipblasHandle_t &handle, const thrust::device_vector<float> &matrix, const unsigned int N, const unsigned int M, const int axis) {
    float alpha = 1.f / N;
    return reduce_alpha(handle, matrix, N, M, alpha, axis);
}


thrust::device_vector<float> reduce_sum(hipblasHandle_t &handle, const thrust::device_vector<float> &matrix, const unsigned int N, const unsigned int M, const int axis) {
    float alpha = 1.f;
    return reduce_alpha(handle, matrix, N, M, alpha, axis);
}

thrust::device_vector<float> do_tsne(hipblasHandle_t &handle, thrust::device_vector<float> &points, const unsigned int N, const unsigned int NDIMS) {
    thrust::device_vector<float> sigmas(N, 1.0f);
    auto pij = compute_pij(handle, points, sigmas, N, NDIMS);
    thrust::device_vector<float> forces(N * PROJDIM);
    thrust::device_vector<float> ys(N * PROJDIM);
    thrust::transform(ys.begin(), ys.end(), ys.begin(), prg(-3.0f, 3.0f));
    thrust::device_vector<float> qij(N * N);
    thrust::device_vector<float> dist(N * N);
    float eta = 1e-10f;
    float loss;
    for (int i = 0; i < 1000; i++) {
        loss = compute_gradients(handle, forces, dist, ys, pij, qij, N, eta);
        thrust::transform(ys.begin(), ys.end(), forces.begin(), ys.begin(), thrust::plus<float>());
        printf("Iteration %d, Loss: %0.2f\n", i, loss);
        if (i > 5)
            break;
    }
    return ys;
}

int main(int argc, char **argv) {
    const unsigned int NDIMS = 50;
    const unsigned int N = 1 << 11;
    
    thrust::default_random_engine rng;
    thrust::uniform_int_distribution<int> dist(10, 99);

    // --- Matrices allocation and initialization
    thrust::device_vector<float> d_X(NDIMS * N);
    for (size_t i = 0; i < d_X.size(); i++) 
        d_X[i] = (float) dist(rng);

    thrust::device_vector<float> sigmas(N, 1.0f);
    hipblasHandle_t handle;
    cublasSafeCall(hipblasCreate(&handle));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    printf("Starting pairwise distance calculation with %u points.\n", N);
    hipEventRecord(start);
    do_tsne(handle, d_X, N, NDIMS);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Elapsed time: %f (ms)\n", milliseconds);
}

