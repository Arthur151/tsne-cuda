#include "hip/hip_runtime.h"
/**
 * @brief Implementation of different distances
 * 
 * @file distance_utils.cu
 * @author David Chan
 * @date 2018-04-04
 */

 #include "util/distance_utils.h"

struct func_sqrt {
    __host__ __device__ float operator()(const float &x) const { return pow(x, 0.5); }
};

// This really does a simultaneous row/col matrix vector broadcast to compute ||x^2|| + ||y^2|| - 2 x^Ty.
// Added fabs to deal with numerical instabilities. I think this is a reasonable solution
 __global__ void assemble_final_result(const float * __restrict__ d_norms_x_2, 
                                       float * __restrict__ d_dots,
                                       const int N)
    {
        const int i = threadIdx.x + blockIdx.x * blockDim.x;
        const int j = threadIdx.y + blockIdx.y * blockDim.y;

        if ((i < N) && (j < N))
            d_dots[i * N + j] = fabs(d_norms_x_2[j] + d_norms_x_2[i] - 2 * d_dots[i * N + j]);
    }
// Code from https://github.com/OrangeOwlSolutions/cuBLAS/blob/master/All_pairs_distances.cu
// Expects N x NDIMS matrix in points
// Squared norms taken from diagnoal of dot product which should be faster and result in actually zeroing out the diagonal in assemble_final_result
void squared_pairwise_dist(hipblasHandle_t &handle, 
                   thrust::device_vector<float> &distances, 
                   const thrust::device_vector<float> &points, 
                   const unsigned int N, 
                   const unsigned int NDIMS) 
{
    const unsigned int BLOCKSIZE = 16;
    // thrust::device_vector<float> squared_vals(points.size());
    // square(points, squared_vals);
    // auto squared_norms = reduce_sum(handle, squared_vals, N, NDIMS, 1);
    
    float alpha = 1.f;
    float beta = 0.f;
    // Could replace this with hipblasSsyrk, might be faster?
	cublasSafeCall(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, NDIMS, &alpha,
		                       thrust::raw_pointer_cast(points.data()), N, thrust::raw_pointer_cast(points.data()), N, &beta,
							   thrust::raw_pointer_cast(distances.data()), N));
  
    typedef thrust::device_vector<float>::iterator Iterator;
    strided_range<Iterator> diag(distances.begin(), distances.end(), N + 1);
    thrust::device_vector<float> squared_norms(N);
    thrust::copy(diag.begin(), diag.end(), squared_norms.begin());

	dim3 dimBlock(BLOCKSIZE, BLOCKSIZE);
	dim3 dimGrid(iDivUp(N, BLOCKSIZE), iDivUp(N, BLOCKSIZE));
	assemble_final_result<<<dimGrid, dimBlock>>>(thrust::raw_pointer_cast(squared_norms.data()), 
                                                 thrust::raw_pointer_cast(distances.data()), N);
                                                 
}

void pairwise_dist(hipblasHandle_t &handle, 
                   thrust::device_vector<float> &distances, 
                   const thrust::device_vector<float> &points, 
                   const unsigned int N, 
                   const unsigned int NDIMS) 
{
    squared_pairwise_dist(handle, distances, points, N, NDIMS);
    sqrt(distances, distances);
}
