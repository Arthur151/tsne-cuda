#include "hip/hip_runtime.h"
/**
 * @brief 
 * 
 * @file math_utils.cu
 * @author David Chan
 * @date 2018-04-04
 */

 #include "util/math_utils.h"

 struct func_square {
    __host__ __device__ float operator()(const float &x) const { return x * x; }
};
struct func_sqrt {
    __host__ __device__ float operator()(const float &x) const { return pow(x, 0.5); }
};
struct func_abs {
    __host__ __device__ float operator()(const float &x) const { return fabs(x); }
};
struct func_nan_or_inf {
    __host__ __device__ bool operator()(const float &x) const { return isnan(x) || isinf(x); }
};

void Math::gauss_normalize(hipblasHandle_t &handle, thrust::device_vector<float> &points, const unsigned int N, const unsigned int NDIMS) {
    auto means = Reduce::reduce_mean(handle, points, N, NDIMS, 0);

    // zero center
    Broadcast::broadcast_matrix_vector(points, means, N, NDIMS, thrust::minus<float>(), 1, 1.f);
    
    // compute standard deviation
    thrust::device_vector<float> squared_vals(points.size());
    Math::square(points, squared_vals);
    auto norm_sum_of_squares = Reduce::reduce_alpha(handle, squared_vals, N, NDIMS, 1.f / (N - 1), 0);
    thrust::device_vector<float> stddev(norm_sum_of_squares.size());
    Math::sqrt(norm_sum_of_squares, stddev);

    // normalize
    Broadcast::broadcast_matrix_vector(points, stddev, N, NDIMS, thrust::divides<float>(), 1, 1.f);
}

void Math::square(const thrust::device_vector<float> &vec, thrust::device_vector<float> &out) {
    thrust::transform(vec.begin(), vec.end(), out.begin(), func_square());
}

void Math::sqrt(const thrust::device_vector<float> &vec, thrust::device_vector<float> &out) {
    thrust::transform(vec.begin(), vec.end(), out.begin(), func_sqrt());
}

float Math::norm(const thrust::device_vector<float> &vec) {
    return std::sqrt( thrust::transform_reduce(vec.begin(), vec.end(), func_square(), 0.0f, thrust::plus<float>()) );
}

bool Math::any_nan_or_inf(const thrust::device_vector<float> &vec) {
    return thrust::transform_reduce(vec.begin(), vec.end(), func_nan_or_inf(), 0, thrust::plus<bool>());
}

void Math::max_norm(thrust::device_vector<float> &vec) {
    float max_val = thrust::transform_reduce(vec.begin(), vec.end(), func_abs(), 0.0f, thrust::maximum<float>());
    thrust::constant_iterator<float> div_iter(max_val);
    thrust::transform(vec.begin(), vec.end(), div_iter, vec.begin(), thrust::divides<float>());
}
