
// Implementation file for the python extensions

#include "ext/pymodule_ext.h"

void pymodule_e_dist(float *points, float *dist, ssize_t *dims) {
    
    // Extract the dimensions of the points array
    ssize_t N_POINTS = dims[0];
    ssize_t N_DIMS = dims[1];

    // Construct device arrays
    thrust::device_vector<float> d_points(N_POINTS*N_DIMS);
    thrust::device_vector<float> d_distances(N_POINTS*N_POINTS);

    // Copy the points to the GPU using thrust
    thrust::copy(points, points+N_DIMS*N_POINTS, d_points.begin());

    // Create the handle, and construct the points
    hipblasHandle_t handle;
    cublasSafeCall(hipblasCreate(&handle));
    pairwise_dist(handle, d_distances, d_points, N_POINTS, N_DIMS);

    // Copy the data back to the CPU
    thrust::copy(d_distances.begin(), d_distances.end(), dist);
}

