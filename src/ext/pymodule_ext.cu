
// Implementation file for the python extensions

#include "ext/pymodule_ext.h"

void pymodule_e_dist(float *points, float *dist, ssize_t *dims) {
    
    // Extract the dimensions of the points array
    ssize_t N_POINTS = dims[0];
    ssize_t N_DIMS = dims[1];

    // Construct device arrays
    thrust::device_vector<float> d_points(N_POINTS*N_DIMS);
    thrust::device_vector<float> d_distances(N_POINTS*N_POINTS);

    // Copy the points to the GPU using thrust
    thrust::copy(points, points+N_DIMS*N_POINTS, d_points.begin());

    // Create the handle, and construct the points
    hipblasHandle_t handle;
    cublasSafeCall(hipblasCreate(&handle));
    Distance::pairwise_dist(handle, d_distances, d_points, N_POINTS, N_DIMS);

    // Copy the data back to the CPU
    thrust::copy(d_distances.begin(), d_distances.end(), dist);
}

void pymodule_naive_tsne(float *points, float *result, ssize_t *dims, int proj_dim, float learning_rate, float perplexity) {
    
    // Extract the dimensions of the points array
    ssize_t N_POINTS = dims[0];
    ssize_t N_DIMS = dims[1];

    // Construct device arrays
    thrust::device_vector<float> d_points(N_POINTS*N_DIMS);

    // Copy the points to the GPU using thrust
    thrust::copy(points, points+N_DIMS*N_POINTS, d_points.begin());

    // Construct the sigmas
    thrust::device_vector<float> sigmas(N_POINTS, 1.0f);

    /*
        RIGHT NOW, WE IGNORE THE PROJECTED DIMENSION, THE LEARNING RATE, and the PERPLEXITY
    */

    // Create the CUBLAS handle
    hipblasHandle_t handle;
    cublasSafeCall(hipblasCreate(&handle));

    // Do the T-SNE
    auto tsne_result = NaiveTSNE::tsne(handle, d_points, N_POINTS, N_DIMS, proj_dim);

    // Copy the data back to the CPU
    thrust::copy(tsne_result.begin(), tsne_result.end(), result);
}

void pymodule_compute_pij(float *points, float* sigmas, float *result, ssize_t *dims) {

     // Extract the dimensions of the points array
     ssize_t N_POINTS = dims[0];
     ssize_t N_DIMS = dims[1];
 
     // Construct device arrays
     thrust::device_vector<float> d_points(N_POINTS*N_DIMS);
 
     // Copy the points to the GPU using thrust
     thrust::copy(points, points+N_DIMS*N_POINTS, d_points.begin());
 
     // Construct the sigmas
     thrust::device_vector<float> d_sigmas(N_POINTS);
     thrust::copy(sigmas, sigmas+N_POINTS, d_sigmas.begin());
 
     // Create the CUBLAS handle
     hipblasHandle_t handle;
     cublasSafeCall(hipblasCreate(&handle));
 
     // Do the T-SNE
     thrust::device_vector<float> pij(N_POINTS*N_POINTS);
     NaiveTSNE::compute_pij(handle, pij, d_points, d_sigmas, N_POINTS, N_DIMS);
 
     // Copy the data back to the CPU
     thrust::copy(pij.begin(), pij.end(), result);

}
