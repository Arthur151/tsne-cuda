
// Implementation file for the python extensions

#include "ext/pymodule_ext.h"

void pymodule_e_dist(float *points, float *dist, ssize_t *dims) {
    
    // Extract the dimensions of the points array
    ssize_t N_POINTS = dims[0];
    ssize_t N_DIMS = dims[1];

    // Construct device arrays
    thrust::device_vector<float> d_points(N_POINTS*N_DIMS);
    thrust::device_vector<float> d_distances(N_POINTS*N_POINTS);

    // Copy the points to the GPU using thrust
    thrust::copy(points, points+N_DIMS*N_POINTS, d_points.begin());

    // Create the handle, and construct the points
    hipblasHandle_t handle;
    cublasSafeCall(hipblasCreate(&handle));
    Distance::pairwise_dist(handle, d_distances, d_points, N_POINTS, N_DIMS);

    // Copy the data back to the CPU
    thrust::copy(d_distances.begin(), d_distances.end(), dist);
}

void pymodule_naive_tsne(float *points, float *result, ssize_t *dims, int proj_dim, float perplexity, float early_ex, 
                            float learning_rate, int n_iter,  int n_iter_np, float min_g_norm) {
    
    // Extract the dimensions of the points array
    ssize_t N_POINTS = dims[0];
    ssize_t N_DIMS = dims[1];

    // Construct device arrays
    thrust::device_vector<float> d_points(N_POINTS*N_DIMS);

    // Copy the points to the GPU using thrust
    thrust::copy(points, points+N_DIMS*N_POINTS, d_points.begin());

    // Construct the sigmas
    thrust::device_vector<float> sigmas(N_POINTS, 1.0f);

    // Create the CUBLAS handle
    hipblasHandle_t handle;
    cublasSafeCall(hipblasCreate(&handle));

    // Do the T-SNE
    auto tsne_result = NaiveTSNE::tsne(handle, d_points, N_POINTS, N_DIMS, proj_dim, perplexity, 
                                            early_ex, learning_rate, n_iter, n_iter_np, min_g_norm);

    // Copy the data back to the CPU
    thrust::copy(tsne_result.begin(), tsne_result.end(), result);
}

void pymodule_compute_pij(float *points, float* sigmas, float *result, ssize_t *dims) {

     // Extract the dimensions of the points array
     ssize_t N_POINTS = dims[0];
     ssize_t N_DIMS = dims[1];
 
     // Construct device arrays
     thrust::device_vector<float> d_points(N_POINTS*N_DIMS);
 
     // Copy the points to the GPU using thrust
     thrust::copy(points, points+N_DIMS*N_POINTS, d_points.begin());
 
     // Construct the sigmas
     thrust::device_vector<float> d_sigmas(N_POINTS);
     thrust::copy(sigmas, sigmas+N_POINTS, d_sigmas.begin());
 
     // Create the CUBLAS handle
     hipblasHandle_t handle;
     cublasSafeCall(hipblasCreate(&handle));
 
     // Do the T-SNE
     thrust::device_vector<float> pij(N_POINTS*N_POINTS);
     NaiveTSNE::compute_pij(handle, pij, d_points, d_sigmas, N_POINTS, N_DIMS);
 
     // Copy the data back to the CPU
     thrust::copy(pij.begin(), pij.end(), result);

}

thrust::device_vector<float> tsne(hipblasHandle_t &dense_handle, 
    hipsparseHandle_t &sparse_handle,
      float* points, 
      unsigned int N_POINTS, 
      unsigned int N_DIMS, 
      unsigned int PROJDIM, 
      float perplexity, 
      float early_ex, 
      float learning_rate, 
      unsigned int n_iter, 
      unsigned int n_iter_np, 
      float min_g_norm);

void pymodule_bh_tsne(float *points, float *result, ssize_t *dims, int proj_dim, float perplexity, float early_ex, 
    float learning_rate, int n_iter,  int n_iter_np, float min_g_norm) {

    // Extract the dimensions of the points array
    ssize_t N_POINTS = dims[0];
    ssize_t N_DIMS = dims[1];

    // Create the CUBLAS handles
    hipblasHandle_t dense_handle;
    cublasSafeCall(hipblasCreate(&dense_handle));
    hipsparseHandle_t sparse_handle;
    cusparseSafeCall(hipsparseCreate(&sparse_handle));

    // Do the t-SNE
    thrust::device_vector<float> tsne_results = BHTSNE::tsne(dense_handle, sparse_handle, points, 
                                                              N_POINTS, N_DIMS, 2, perplexity, early_ex, learning_rate, 
                                                              n_iter, n_iter_np, min_g_norm, false, false, 5.0, 0, 1023, "tcp://localhost:5556", nullptr);

    // Copy the data back from the GPU
    thrust::copy(tsne_results.begin(), tsne_results.end(), result);
}