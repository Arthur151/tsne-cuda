#include "hip/hip_runtime.h"
#include "common.h"

std::vector<float> squared_pairwise_dist(std::vector<float> &points, const unsigned int N, const unsigned int NDIMS) {
	std::vector<float> squared_pairwise_dist(N * N, 0);
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			for(int k = 0; k < NDIMS; k++) {
				squared_pairwise_dist[i * N + j] += (points[i * NDIMS + k] - points[j * NDIMS + k]) * (points[i * NDIMS + k] - points[j * NDIMS + k]); 
			}
		}
    }
	return squared_pairwise_dist;
}



bool perplexity_equal(const float delta, float perplexity, float target_perplexity) {
	return (perplexity >= target_perplexity - delta) && (perplexity <= target_perplexity + delta);
}


float get_perplexity(std::vector<float> & pij, const unsigned int i, unsigned int N) {
	float entropy = 0.0f;
	for (int j = 0; j < N; j++) {
		if (j != i) {
			entropy += pij[i*N + j] * std::log2(pij[i*N + j]);
		}
	}
	return std::pow(2, -entropy);
}

bool compare_perplexity(std::vector<float>& pij, 
					   float& lo, 
					   float& mid, 
					   float& hi, 
					   const unsigned int i, 
					   const unsigned int N, 
					   const float delta, 
					   const float target_perplexity) {
	float perplexity = get_perplexity(pij, i, N);
	if (perplexity_equal(delta, perplexity, target_perplexity)) {
		return true;
	} else if (perplexity > target_perplexity) {
		hi = mid - delta;
	} else {
		lo = mid + delta;
	}

	mid = (lo + hi)/2;
	return false;
}


void recompute_pij_row_cpu(std::vector<float> &points, 
	                           std::vector<float> &pij, 
	                           float sigma,
	                           float i, 
	                           const unsigned int N, 
	                           const unsigned int NDIMS) {
	std::vector<float> dists = squared_pairwise_dist(points, N, NDIMS);
	for (int j = 0; j < N; j++) {
		float denom = 0;
		for (int k = 0; k < N; k++) {
			if (k != i) {
				denom += std::exp(-(dists[i * N + k] / (2 * sigma * sigma)));
			}
		}
        if (i != j) {
		    pij[i * N + j] = std::exp(-dists[i * N + j] / (2 * sigma * sigma)) / denom;
        }

    }
}

std::vector<float> compute_pij_cpu(std::vector<float> &points, 
	                           std::vector<float> &sigma, 
	                           const unsigned int N, 
	                           const unsigned int NDIMS) {
	std::vector<float> pij_out(N * N, 0.0f);
	std::vector<float> dists = squared_pairwise_dist(points, N, NDIMS);

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			float denom = 0;
			for (int k = 0; k < N; k++) {
				if (k != i) {
					denom += std::exp(-(dists[i * N + k] / (2 * sigma[i] * sigma[i])));
				}
			}
            if (i != j) {
			    pij_out[i * N + j] = std::exp(-dists[i * N + j] / (2 * sigma[i] * sigma[i])) / denom;
            }
           
        }
	}
	return pij_out;

}

std::vector<float> compute_qij_cpu(std::vector<float>& ys, const unsigned int N, const unsigned int PROJDIMS) {
	
	std::vector<float> qij_out(N * N);
	std::vector<float> dists = squared_pairwise_dist(ys, N, PROJDIMS);

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			float denom = 0;
			for (int k = 0; k < N; k++) {
				if (k != i) {
					denom += 1 / (1 + (dists[i * N + k]));
				}
			}
			qij_out[i * N + j] = (1 / (1 + dists[i * N + j])) / denom;
		}
	}
	return qij_out;
}

float kl_div(float pij, float qij) {
	return pij * std::log(pij / qij);
}

float compute_gradients_cpu(std::vector<float> &forces,
                    	std::vector<float> &dist, 
                        std::vector<float> &ys, 
                        std::vector<float> &pij, 
                        std::vector<float> &qij, 
                        const unsigned int N,
                        float eta) {

	float loss = 0.0f;
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			float pij_sym = (pij[i * N + j] + pij[j * N + i]) / (2 * N);
			float qij_sym = (qij[i * N + j] + qij[j * N + i]) / (2 * N);
			loss += kl_div(pij_sym, qij_sym);
		}

	}
	return loss;

}


std::vector<float> sigmas_search_cpu(std::vector<float> &points,  
	                           const unsigned int N, 
	                           const unsigned int NDIMS,
	                           float target_perplexity) {
	const float max_sigma = 1000.0f;
	const float delta = 0.1f;
	std::vector<float> sigmas(N, max_sigma/2);
	std::vector<float> pij = compute_pij_cpu(points, sigmas, N, NDIMS);
	for (int i = 0; i < N; i++) {
		bool found = false;
		float lo = 0.0f;
		float hi = max_sigma;
		float mid = (lo + hi)/ 2;
		while (!found) {
			found = compare_perplexity(pij, lo, mid, hi, i, N, delta, target_perplexity);
			recompute_pij_row_cpu(points, pij, mid, i, N, NDIMS);
        }
		sigmas[i] = mid;
	}
	return sigmas;

}

std::vector<float> naive_tsne_cpu(std::vector<float> &points, 
                              const unsigned int N, 
                              const unsigned int NDIMS) {
	std::default_random_engine generator;
  	std::uniform_real_distribution<double> distribution(-10.0f,10.0f);
 	const unsigned int NPROJDIM = 2;
 	std::vector<float> ys(N * NPROJDIM);
 	for (int i = 0; i < N * NPROJDIM; i++) {
 		ys[i] = distribution(generator);
 	}
	for (int i = 0; i < 1000; i++) {

	}
    return ys;

}
